#include <iostream>
#include <hip/hip_runtime.h>

__global__ void addVectors(int *a, int *b, int *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    const int arraySize = 5;
    const int arrayBytes = arraySize * sizeof(int);

    // Host arrays
    int h_a[arraySize] = {1, 2, 3, 4, 5};
    int h_b[arraySize] = {10, 20, 30, 40, 50};
    int h_c[arraySize];

    // Device arrays
    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, arrayBytes);
    hipMalloc((void**)&d_b, arrayBytes);
    hipMalloc((void**)&d_c, arrayBytes);

    // Copy data from host to device
    hipMemcpy(d_a, h_a, arrayBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, arrayBytes, hipMemcpyHostToDevice);

    // Launch kernel
    int blockSize = 256;
    int numBlocks = (arraySize + blockSize - 1) / blockSize;
    addVectors<<<numBlocks, blockSize>>>(d_a, d_b, d_c, arraySize);

    // Copy result back to host
    hipMemcpy(h_c, d_c, arrayBytes, hipMemcpyDeviceToHost);

    // Print result
    std::cout << "Result: ";
    for (int i = 0; i < arraySize; i++) {
        std::cout << h_c[i] << " ";
    }
    std::cout << std::endl;

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}