#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <hip/hip_runtime.h>

#include "optixparams.cuh"
#include "cudamath.cuh"
#include "sampling.cuh"

struct Ray {
    float3 origin;
    float3 direction;
};

__device__ inline Ray makeCameraRay(const float2& uv) {
    const float4 origin = params.clipToWorld[3]; // = params.clipToWorld * make_float4(0.0f, 0.0f, 0.0f, 1.0f);
    const float4 clipTarget = make_float4(-2.0f * uv + 1.0f, -1.0f, 1.0f);
    const float4 target = params.clipToWorld * clipTarget;
    const float3 origin3 = make_float3(origin) / origin.w;
    const float3 dir3 = normalize(origin3 - make_float3(target) / target.w);
    return {origin3, dir3};
}

__device__ inline float getRand(uint depth, uint offset, float rotation) {
    return fract(getRand(depth, offset) + rotation);
}

__device__ inline float2 getRand(uint depth, uint offset, float r0, float r1) {
    return fract(make_float2(getRand(depth, offset + 0) + r0, getRand(depth, offset + 1) + r1));
}

__device__ inline float3 getRand(uint depth, uint offset, float r0, float r1, float r2) {
    return fract(make_float3(getRand(depth, offset + 0) + r0, getRand(depth, offset + 1) + r1, getRand(depth, offset + 2) + r2));
}

__device__ inline void setBaseColor(const float3& value) {
    optixSetPayload_0(__float_as_uint(value.x));
    optixSetPayload_1(__float_as_uint(value.y));
    optixSetPayload_2(__float_as_uint(value.z));
}

__device__ inline void setNormal(const float3& value) {
    optixSetPayload_3(__float_as_uint(value.x));
    optixSetPayload_4(__float_as_uint(value.y));
    optixSetPayload_5(__float_as_uint(value.z));
}

__device__ inline void setTangent(const float3& value) {
    optixSetPayload_6(__float_as_uint(value.x));
    optixSetPayload_7(__float_as_uint(value.y));
    optixSetPayload_8(__float_as_uint(value.z));
}

__device__ inline void setEmission(const float3& value) {
    optixSetPayload_9(__float_as_uint(value.x));
    optixSetPayload_10(__float_as_uint(value.y));
    optixSetPayload_11(__float_as_uint(value.z));
}

__device__ inline void setRoughness(const float value) {
    optixSetPayload_12(__float_as_uint(value));
}

__device__ inline void setMetallic(const float value) {
    optixSetPayload_13(__float_as_uint(value));
}

__device__ inline void setTransmission(const float value) {
    optixSetPayload_14(__float_as_uint(value));
}

__device__ inline void setArea(const float value) {
    optixSetPayload_15(__float_as_uint(value));
}

__device__ inline void setT(const float value) {
    optixSetPayload_16(__float_as_uint(value));
}

__device__ constexpr inline Payload getPayload(const std::array<uint, 17>& values) {
    return {
        make_float3(__uint_as_float(values[0]), __uint_as_float(values[1]), __uint_as_float(values[2])),
        make_float3(__uint_as_float(values[3]), __uint_as_float(values[4]), __uint_as_float(values[5])),
        make_float3(__uint_as_float(values[6]), __uint_as_float(values[7]), __uint_as_float(values[8])),
        make_float3(__uint_as_float(values[9]), __uint_as_float(values[10]), __uint_as_float(values[11])),
        __uint_as_float(values[12]),
        __uint_as_float(values[13]),
        __uint_as_float(values[14]),
        __uint_as_float(values[15]),
        __uint_as_float(values[16]),
    };
}

__device__ inline Payload trace(const Ray& ray) {
    uint a, b, c, d, e, f, g, h, i, j, k, l, m, n, o, p, q;
    optixTraverse(
        params.handle,
        ray.origin, ray.direction,
        0.0f, MAX_T, // tmin, tmax
        0.0f, // rayTime
        OptixVisibilityMask(255), OPTIX_RAY_FLAG_NONE,
        0, 1, 0, // SBT offset, stride, miss index
        a, b, c, d, e, f, g, h, i, j, k, l, m, n, o, p, q // Payload
    );
    //const auto data = reinterpret_cast<HitData*>(optixGetSbtDataPointer());
    //optixReorder(data->materialID, 3); // TODO: Provide coherence hints
    optixInvoke(a, b, c, d, e, f, g, h, i, j, k, l, m, n, o, p, q);
    return getPayload({a, b, c, d, e, f, g, h, i, j, k, l, m, n, o, p, q});
}

__device__ inline bool traceOcclusion(const float3& a, const float3& b) {
    const auto dir = b - a;
    optixTraverse(
        params.handle,
        a, dir,
        0.0f, 1.0f, // tmin, tmax
        0.0f, // rayTime
        OptixVisibilityMask(255), OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT | OPTIX_RAY_FLAG_DISABLE_ANYHIT,
        0, 1, 0 // SBT offset, stride, miss index
    );
    return optixHitObjectIsHit();
}

__device__ inline NRCInput encodeInput(const float3& position, const float3& wo, const float3& wn, const float3& diffuse, const float3& specular, float alpha) {
    return {
        .position = position * 0.1f + 0.5f, // TODO: Normalize position
        .wo = toNormSpherical(wo), // Switch to Octahedral
        .wn = toNormSpherical(wn), // Switch to Octahedral
        //.roughness = 1 - exp(-alpha),
        .roughness = alpha,
        .diffuse = diffuse,
        .specular = specular, // directional albedo FDG
    };
}

__device__ inline void pushNRCInput(float* to, const NRCInput& input) {
    to[0] = input.position.x;
    to[1] = input.position.y;
    to[2] = input.position.z;
    to[3] = input.wo.x;
    to[4] = input.wo.y;
    to[5] = input.wn.x;
    to[6] = input.wn.y;
    to[7] = input.roughness;
    to[8] = input.diffuse.x;
    to[9] = input.diffuse.y;
    to[10] = input.diffuse.z;
    to[11] = input.specular.x;
    to[12] = input.specular.y;
    to[13] = input.specular.z;
}

__device__ inline void pushNRCTrainInput(const NRCInput& input) {
    const auto i = atomicAdd(params.trainingIndexPtr, 1u);
    pushNRCInput(params.trainingInput + ((i + 1) % NRC_BATCH_SIZE) * NRC_INPUT_SIZE, input);
}

__device__ inline void pushNRCOutput(float* to, const NRCOutput& output) {
    to[0] = output.radiance.x;
    to[1] = output.radiance.y;
    to[2] = output.radiance.z;
}

extern "C" __global__ void __raygen__rg() {
    const auto idx = optixGetLaunchIndex();
    const auto dim = optixGetLaunchDimensions();
    const auto i = idx.y * params.dim.x + idx.x;
    const auto rotation = params.rotationTable[i];

    const auto jitter = fract(make_float2(getRand(0), getRand(1)) + make_float2(rotation));
    const auto uv = (make_float2(idx.x, idx.y) + jitter) / make_float2(dim.x, dim.y);
    auto ray = makeCameraRay(uv);

    const auto nee = params.lightTable && (params.flags & NEE_FLAG);

    Payload payload;
    auto color = make_float3(0.0f);
    auto throughput = make_float3(1.0f);
    auto prevBrdfPdf = 1.0f;
    auto diracEvent = true;

    auto trainDepth = -1;
    // TODO: Train the whole path
    // TODO: Use stratified sampling
    if (getRand(1, 2, rotation.y) < NRC_BATCH_SIZE / float(params.dim.x * params.dim.y)) {
        trainDepth = int(getRand(1, 3, rotation.z) * 6) + 1;
    }
    auto trainTarget = NRCOutput{};
    auto reflectanceFactorizationTerm = make_float3(1.0f);
    auto trainThroughput = make_float3(1.0f);
    bool isTrainingPath = trainDepth >= 0;
    bool writeTrainingSample = false;
    float3 inferenceThroughput = make_float3(1.0f);
    float3 inferencePlus = make_float3(0.0f);

    NRCInput nrcQuery {};
    
    for (uint depth = 1; depth < MAX_BOUNCES; depth++) {
        if (depth == trainDepth) {
            trainTarget.radiance = make_float3(0.0f);
            trainThroughput = make_float3(1.0f);
        }

        // Russian roulette
        const float pContinue = min(luminance(throughput) * params.russianRouletteWeight, 1.0f);
        if (getRand(depth, 3, rotation.z) >= pContinue) break;
        throughput /= pContinue;
        trainThroughput /= pContinue;

        payload = trace(ray);

        if (isinf(payload.t)) {
            color += throughput * payload.emission;
            trainTarget.radiance += trainThroughput * payload.emission;
            break; // Skybox
        }

        auto n = payload.normal;
        const auto wo = -ray.direction;
        const auto inside = dot(n, wo) < 0.0f;
        n = inside ? -n : n;

        if (luminance(payload.emission) > 0.0f) {
            auto weight = 1.0f;
            if (nee && !diracEvent) {
                // NOTE: Maybe calculating the prevBrdfPdf here only when necessary is faster
                const auto lightPdf = lightPdfUniform(wo, payload.t, n, payload.area);
                weight = balanceHeuristic(prevBrdfPdf, lightPdf);
#ifdef DEBUGPRINT
    printf("Weight: %.3f BRDF: %.3f Light: %.3f\n", weight, prevBrdfPdf, lightPdf);
#endif
            }
            color += throughput * payload.emission * weight;
            trainTarget.radiance += trainThroughput * payload.emission * weight;
        }

        const auto hitPoint = ray.origin + payload.t * ray.direction;
        const auto alpha = payload.roughness * payload.roughness;
        const auto metallic = payload.metallic;
        const auto baseColor = payload.baseColor; // baseColor

        if (depth == trainDepth && luminance(payload.emission) < 1e-3f) { // NOTE: Skipping emissive vertices reduces variance
            const auto F0 = mix(make_float3(0.04f), baseColor, metallic);
            const auto albedo = (1.0f - metallic) * baseColor;
            auto trainInput = encodeInput(hitPoint, wo, n, albedo, F0, alpha);
            reflectanceFactorizationTerm = 1.0f / max(trainInput.diffuse + trainInput.specular, 1e-3f);
            const auto inputIdx = (i % NRC_BATCH_SIZE) * NRC_INPUT_SIZE;
            pushNRCInput(params.trainingInput + inputIdx, trainInput);
            //pushNRCTrainInput(trainInput);
            writeTrainingSample = true;
        }

        // NRC Inference Input
        if ((params.inferenceMode == InferenceMode::FIRST_VERTEX || params.inferenceMode == InferenceMode::RAW_CACHE) && depth == 1) {
            const auto F0 = mix(make_float3(0.04f), baseColor, metallic);
            const auto albedo = (1.0f - metallic) * baseColor;
            nrcQuery = encodeInput(hitPoint, wo, n, albedo, F0, alpha);
            inferenceThroughput = throughput;
            inferencePlus = color;
            if (!isTrainingPath) break;
        }

        // Next event estimation
        // TODO: Dirac check
        if (nee) {
            const auto sample = sampleLight(getRand(depth, 0, rotation.w, rotation.x, rotation.y), hitPoint);
            const auto cosThetaS = dot(sample.wi, n);
            //if (abs(cosThetaS) > 0.0f && abs(sample.cosThetaL) > 0.0f) {
                const auto brdf = evalDisney(wo, sample.wi, n, baseColor, metallic, alpha, payload.transmission, inside);
                const auto surfacePoint = hitPoint + n * copysignf(params.sceneEpsilon, cosThetaS);
                const auto lightPoint = sample.position - sample.n * copysignf(params.sceneEpsilon, dot(sample.wi, sample.n));
                if (!brdf.isDirac && brdf.pdf > 0.0f && !traceOcclusion(surfacePoint, lightPoint)) {
                    const auto weight = balanceHeuristic(sample.pdf, brdf.pdf);
                    color += throughput * brdf.throughput * sample.emission * weight / sample.pdf;
                    trainTarget.radiance += trainThroughput * brdf.throughput * sample.emission * weight / sample.pdf;
#ifdef DEBUGPRINT
    if (getRand(depth, 0, rotation.y) < 0.001f) printf("\t\t\t\t\t\tNEE We: %.3f BRDF: %.3f Light: %.3f\n", weight, brdf.pdf, sample.pdf);
#endif
                }
            //}
        }

        // NRC Inference Input
        if (params.inferenceMode == InferenceMode::FIRST_VERTEX_WITH_NEE && depth == 1) {
            const auto F0 = mix(make_float3(0.04f), baseColor, metallic);
            const auto albedo = (1.0f - metallic) * baseColor;
            nrcQuery = encodeInput(hitPoint, wo, n, albedo, F0, alpha);
            inferenceThroughput = throughput;
            inferencePlus = color;
            if (!isTrainingPath) break;
        }

        // TODO: Move sampling into closesthit to benefit from reordering
        const auto sample = sampleDisney(getRand(depth, 0, rotation.w), getRand(depth, 1, rotation.x, rotation.y), getRand(depth, 1, rotation.z, rotation.w), wo, n, inside, baseColor, metallic, alpha, payload.transmission);
        
        ray = Ray{hitPoint + n * copysignf(params.sceneEpsilon, dot(sample.direction, n)), sample.direction};
        throughput *= sample.throughput;
        trainThroughput *= sample.throughput;
        prevBrdfPdf = sample.pdf;
        diracEvent = sample.isDirac;

        // NRC Inference Input
        if (params.inferenceMode == InferenceMode::FIRST_DIFFUSE && !sample.isSpecular) {
            const auto F0 = mix(make_float3(0.04f), baseColor, metallic);
            const auto albedo = (1.0f - metallic) * baseColor;
            nrcQuery = encodeInput(hitPoint, wo, n, albedo, F0, alpha);
            inferenceThroughput = throughput;
            inferencePlus = color;
            if (!isTrainingPath) break;
        }
    }

    if (writeTrainingSample) {
        trainTarget.radiance *= reflectanceFactorizationTerm;
        const auto outputIdx = (i % NRC_BATCH_SIZE) * NRC_OUTPUT_SIZE;
        pushNRCOutput(params.trainingTarget + outputIdx, trainTarget);
    }

    const auto inputIdx = i * NRC_INPUT_SIZE;
    pushNRCInput(params.inferenceInput + inputIdx, nrcQuery);
    params.inferenceThroughput[i] = inferenceThroughput;

    // NOTE: We should not need to prevent NaNs
    // FIXME: NaNs
    //if (isfinite(color))
    if (params.inferenceMode == InferenceMode::NO_INFERENCE) {
        params.image[i] = mix(params.image[i], make_float4(max(color, 0.0f), 1.0f), params.weight); // FIXME: Negative colors
    } else {
        params.image[i] = mix(params.image[i], make_float4(max(inferencePlus, 0.0f), 1.0f), params.weight); // FIXME: Negative colors
    }
}

extern "C" __global__ void __closesthit__ch() {
    // Get optix built-in variables
    const auto bary2 = optixGetTriangleBarycentrics();
    const auto bary = make_float3(1.0f - bary2.x - bary2.y, bary2);
    const auto data = reinterpret_cast<HitData*>(optixGetSbtDataPointer());

    // Get triangle vertices
    const auto idx = data->indexBuffer[optixGetPrimitiveIndex()];
    const auto v0 = data->vertexData[idx.x];
    const auto v1 = data->vertexData[idx.y];
    const auto v2 = data->vertexData[idx.z];

    const auto e0 = v1.position - v0.position;
    const auto e1 = v2.position - v0.position;
    const auto area = 0.5f * length(optixTransformVectorFromObjectToWorldSpace(cross(e0, e1)));

    // Interpolate normal
    const auto objectSpaceNormal = bary.x * v0.normal + bary.y * v1.normal + bary.z * v2.normal;

    // Interpolate tangent
    const auto objectSpaceTangentWithOrientation = bary.x * v0.tangent + bary.y * v1.tangent + bary.z * v2.tangent;
    const auto objectSpaceTangent = make_float3(objectSpaceTangentWithOrientation);
    const auto worldSpaceTangent = optixTransformVectorFromObjectToWorldSpace(objectSpaceTangent);

    const auto texCoord = bary.x * v0.texCoord + bary.y * v1.texCoord + bary.z * v2.texCoord;

    // Get material
    const auto material = params.materials[data->materialID];
    
    auto baseColor = material.baseColor;
    if (material.baseMap) baseColor *= make_float3(tex2D<float4>(material.baseMap, texCoord.x, texCoord.y));

    auto mr = make_float2(material.metallic, material.roughness);
    if (material.mrMap) mr *= make_float2(tex2D<float4>(material.mrMap, texCoord.x, texCoord.y));

    // NOTE: Normal mapping produces artifacts with pathtracing: See Microfacet-based Normal Mapping for Robust Monte Carlo Path Tracing by Schüssler et al. 2017 for a solution
    if (material.normalMap) { // MikkTSpace normal mapping
        const auto tangentOrientation = objectSpaceTangentWithOrientation.w;
        const auto tangentSpaceNormal = make_float3(tex2D<float4>(material.normalMap, texCoord.x, texCoord.y)) * 2.0f - 1.0f;
        const auto objectSpaceBitangent = cross(objectSpaceNormal, objectSpaceTangent) * tangentOrientation;
        setNormal(normalize(optixTransformNormalFromObjectToWorldSpace(tangentSpaceNormal.x * objectSpaceTangent + tangentSpaceNormal.y * objectSpaceBitangent + tangentSpaceNormal.z * objectSpaceNormal)));
    } else {
        setNormal(normalize(optixTransformNormalFromObjectToWorldSpace(objectSpaceNormal)));
    }
    
    setBaseColor(baseColor);
    setTangent(worldSpaceTangent);
    setMetallic(mr.x);
    setEmission(material.emission);
    setRoughness(mr.y);
    setTransmission(params.flags & TRANSMISSION_FLAG ? material.transmission : 0.0f);
    setArea(area);
    setT(optixGetRayTmax());
}

extern "C" __global__ void __miss__ms() {
    const auto dir = optixGetWorldRayDirection();
    auto sky = make_float3(0.03f);

    setEmission(sky);
    setT(INFINITY);
}