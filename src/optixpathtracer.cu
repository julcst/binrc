#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <hip/hip_runtime.h>

#include "optixparams.cuh"
#include "cudamath.cuh"
#include "sampling.cuh"

struct Ray {
    float3 origin;
    float3 direction;
};

__device__ Ray makeCameraRay(const float2& uv) {
    const float4 origin = params.clipToWorld[3]; // = params.clipToWorld * make_float4(0.0f, 0.0f, 0.0f, 1.0f);
    const float4 clipTarget = make_float4(-2.0f * uv + 1.0f, -1.0f, 1.0f);
    const float4 target = params.clipToWorld * clipTarget;
    const float3 origin3 = make_float3(origin) / origin.w;
    const float3 dir3 = normalize(origin3 - make_float3(target) / target.w);
    return Ray{origin3, dir3};
}

__device__ inline float getRand(uint depth, uint offset, float rotation) {
    return fract(getRand(depth, offset) + rotation);
}

__device__ inline float2 getRand(uint depth, uint offset, float r0, float r1) {
    return fract(make_float2(getRand(depth, offset + 0) + r0, getRand(depth, offset + 1) + r1));
}

__device__ inline float3 getRand(uint depth, uint offset, float r0, float r1, float r2) {
    return fract(make_float3(getRand(depth, offset + 0) + r0, getRand(depth, offset + 1) + r1, getRand(depth, offset + 2) + r2));
}

struct Payload {
    float3 baseColor; // Linear RGB base color
    float3 normal; // World space normal, guaranteed to be normalized
    float3 tangent; // World space tenagent, not normalized
    float3 emission; // Linear RGB emission color
    float roughness;
    float metallic;
    float transmission;
    float area;
    float t; // Distance of intersection on ray, set to INFINITY if no intersection
};

__device__ void setBaseColor(const float3& value) {
    optixSetPayload_0(__float_as_uint(value.x));
    optixSetPayload_1(__float_as_uint(value.y));
    optixSetPayload_2(__float_as_uint(value.z));
}

__device__ void setNormal(const float3& value) {
    optixSetPayload_3(__float_as_uint(value.x));
    optixSetPayload_4(__float_as_uint(value.y));
    optixSetPayload_5(__float_as_uint(value.z));
}

__device__ void setTangent(const float3& value) {
    optixSetPayload_6(__float_as_uint(value.x));
    optixSetPayload_7(__float_as_uint(value.y));
    optixSetPayload_8(__float_as_uint(value.z));
}

__device__ void setEmission(const float3& value) {
    optixSetPayload_9(__float_as_uint(value.x));
    optixSetPayload_10(__float_as_uint(value.y));
    optixSetPayload_11(__float_as_uint(value.z));
}

__device__ void setRoughness(const float value) {
    optixSetPayload_12(__float_as_uint(value));
}

__device__ void setMetallic(const float value) {
    optixSetPayload_13(__float_as_uint(value));
}

__device__ void setTransmission(const float value) {
    optixSetPayload_14(__float_as_uint(value));
}

__device__ void setArea(const float value) {
    optixSetPayload_15(__float_as_uint(value));
}

__device__ void setT(const float value) {
    optixSetPayload_16(__float_as_uint(value));
}

__device__ Payload getPayload(uint a, uint b, uint c, uint d, uint e, uint f, uint g, uint h, uint i, uint j, uint k, uint l, uint m, uint n, uint o, uint p, uint q) {
    return Payload{
        make_float3(__uint_as_float(a), __uint_as_float(b), __uint_as_float(c)),
        make_float3(__uint_as_float(d), __uint_as_float(e), __uint_as_float(f)),
        make_float3(__uint_as_float(g), __uint_as_float(h), __uint_as_float(i)),
        make_float3(__uint_as_float(j), __uint_as_float(k), __uint_as_float(l)),
        __uint_as_float(m),
        __uint_as_float(n),
        __uint_as_float(o),
        __uint_as_float(p),
        __uint_as_float(q),
    };
}

__device__ Payload trace(const Ray& ray) {
    uint a, b, c, d, e, f, g, h, i, j, k, l, m, n, o, p, q;
    optixTraverse(
        params.handle,
        ray.origin, ray.direction,
        0.0f, MAX_T, // tmin, tmax
        0.0f, // rayTime
        OptixVisibilityMask(255), OPTIX_RAY_FLAG_NONE,
        0, 1, 0, // SBT offset, stride, miss index
        a, b, c, d, e, f, g, h, i, j, k, l, m, n, o, p, q // Payload
    );
    //const auto data = reinterpret_cast<HitData*>(optixGetSbtDataPointer());
    //optixReorder(data->materialID, 3); // TODO: Provide coherence hints
    optixInvoke(a, b, c, d, e, f, g, h, i, j, k, l, m, n, o, p, q);
    return getPayload(a, b, c, d, e, f, g, h, i, j, k, l, m, n, o, p, q);
}

__device__ bool traceOcclusion(const float3& a, const float3& b) {
    const auto dir = b - a;
    optixTraverse(
        params.handle,
        a, dir,
        0.0f, 1.0f, // tmin, tmax
        0.0f, // rayTime
        OptixVisibilityMask(255), OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT | OPTIX_RAY_FLAG_DISABLE_ANYHIT,
        0, 1, 0 // SBT offset, stride, miss index
    );
    return optixHitObjectIsHit();
}

__device__ void pushTrainingSample(float3 pos, float3 color) {
    const auto idx = optixGetLaunchIndex();
    const auto dim = optixGetLaunchDimensions();
    const auto i = (idx.y * params.dim.x + idx.x) % NRC_BATCH_SIZE;
    const auto inputIdx = i * NRC_INPUT_SIZE;
    const auto outputIdx = i * NRC_OUTPUT_SIZE;
    params.trainingInput[inputIdx + 0] = pos.x;
    params.trainingInput[inputIdx + 1] = pos.y;
    params.trainingInput[inputIdx + 2] = pos.z;
    params.trainingTarget[outputIdx + 0] = color.x;
    params.trainingTarget[outputIdx + 1] = color.y;
    params.trainingTarget[outputIdx + 2] = color.z;
}

extern "C" __global__ void __raygen__rg() {
    const auto idx = optixGetLaunchIndex();
    const auto dim = optixGetLaunchDimensions();
    const auto i = idx.y * params.dim.x + idx.x;
    const auto rotation = params.rotationTable[i];

    const auto jitter = fract(make_float2(getRand(0), getRand(1)) + make_float2(rotation));
    const auto uv = (make_float2(idx.x, idx.y) + jitter) / make_float2(dim.x, dim.y);
    auto ray = makeCameraRay(uv);

    const auto nee = params.lightTable && (params.flags & NEE_FLAG);

    Payload payload;
    auto color = make_float3(0.0f);
    auto throughput = make_float3(1.0f);
    auto prevBrdfPdf = 1.0f;
    auto diracEvent = true;

    auto trainDepth = -1;
    if (getRand(1, 2, rotation.y) < NRC_BATCH_SIZE / float(params.dim.x * params.dim.y)) {
        trainDepth = int(getRand(1, 3, rotation.z) * 6) + 1;
    }
    auto trainInput = NRCInput{};
    auto trainTarget = NRCOutput{};
    auto trainThroughput = make_float3(1.0f);

    auto nrcQuery = NRCInput{};
    
    for (uint depth = 1; depth < MAX_BOUNCES; depth++) {
        if (depth == trainDepth) {
            trainTarget.radiance = make_float3(0.0f);
            trainThroughput = make_float3(1.0f);
        }

        // Russian roulette
        const float pContinue = min(luminance(throughput) * params.russianRouletteWeight, 1.0f);
        if (getRand(depth, 3, rotation.z) >= pContinue) break;
        throughput /= pContinue;
        trainThroughput /= pContinue;

        payload = trace(ray);

        if (isinf(payload.t)) {
            color += throughput * payload.emission;
            trainTarget.radiance += trainThroughput * payload.emission;
            break; // Skybox
        }

        auto n = payload.normal;
        const auto wo = -ray.direction;
        const auto inside = dot(n, wo) < 0.0f;
        n = inside ? -n : n;

        if (luminance(payload.emission) > 0.0f) {
            auto weight = 1.0f;
            if (nee && !diracEvent) {
                // NOTE: Maybe calculating the prevBrdfPdf here only when necessary is faster
                const auto lightPdf = lightPdfUniform(wo, payload.t, n, payload.area);
                weight = balanceHeuristic(prevBrdfPdf, lightPdf);
#ifdef DEBUGPRINT
    printf("Weight: %.3f BRDF: %.3f Light: %.3f\n", weight, prevBrdfPdf, lightPdf);
#endif
            }
            color += throughput * payload.emission * weight;
            trainTarget.radiance += trainThroughput * payload.emission * weight;
        }

        const auto hitPoint = ray.origin + payload.t * ray.direction;
        const auto alpha = payload.roughness * payload.roughness;
        const auto metallic = payload.metallic;
        const auto baseColor = payload.baseColor; // baseColor

        // NRC Inference Input
        if (depth == 1) {
            const auto F0 = mix(make_float3(0.04f), baseColor, metallic);
            const auto albedo = (1.0f - metallic) * baseColor;
            nrcQuery.position = hitPoint;
            nrcQuery.wo = toNormSpherical(wo);
            nrcQuery.wn = toNormSpherical(n);
            nrcQuery.roughness = 1 - exp(-alpha);
            nrcQuery.diffuse = albedo;
            nrcQuery.specular = F0;
        }

        if (depth == trainDepth) {
            const auto F0 = mix(make_float3(0.04f), baseColor, metallic);
            const auto albedo = (1.0f - metallic) * baseColor;
            trainInput.position = hitPoint;
            trainInput.wo = toNormSpherical(wo);
            trainInput.wn = toNormSpherical(n);
            trainInput.roughness = 1 - exp(-alpha);
            trainInput.diffuse = albedo;
            trainInput.specular = F0;
        }

        // Next event estimation
        // TODO: Dirac check
        if (nee) {
            const auto sample = sampleLight(getRand(depth, 0, rotation.w, rotation.x, rotation.y), hitPoint);
            const auto cosThetaS = dot(sample.wi, n);
            //if (abs(cosThetaS) > 0.0f && abs(sample.cosThetaL) > 0.0f) {
                const auto brdf = evalDisney(wo, sample.wi, n, baseColor, metallic, alpha, payload.transmission, inside);
                const auto surfacePoint = hitPoint + n * copysignf(params.sceneEpsilon, cosThetaS);
                const auto lightPoint = sample.position - sample.n * copysignf(params.sceneEpsilon, dot(sample.wi, sample.n));
                if (!brdf.isDirac && brdf.pdf > 0.0f && !traceOcclusion(surfacePoint, lightPoint)) {
                    const auto weight = balanceHeuristic(sample.pdf, brdf.pdf);
                    color += throughput * brdf.throughput * sample.emission * weight / sample.pdf;
                    trainTarget.radiance += trainThroughput * brdf.throughput * sample.emission * weight / sample.pdf;
#ifdef DEBUGPRINT
    if (getRand(depth, 0, rotation.y) < 0.001f) printf("\t\t\t\t\t\tNEE We: %.3f BRDF: %.3f Light: %.3f\n", weight, brdf.pdf, sample.pdf);
#endif
                }
            //}
        }

        // TODO: Move sampling into closesthit to benefit from reordering
        const auto sample = sampleDisney(getRand(depth, 0, rotation.w), getRand(depth, 1, rotation.x, rotation.y), getRand(depth, 1, rotation.z, rotation.w), wo, n, inside, baseColor, metallic, alpha, payload.transmission);
        
        ray = Ray{hitPoint + n * copysignf(params.sceneEpsilon, dot(sample.direction, n)), sample.direction};
        throughput *= sample.throughput;
        trainThroughput *= sample.throughput;
        prevBrdfPdf = sample.pdf;
        diracEvent = sample.isDirac;
    }

    trainTarget.radiance = trainTarget.radiance / max(trainInput.diffuse + trainInput.specular, 1e-3f);
    
    // NRC Training
    if (isfinite(trainInput.position)
        && isfinite(trainInput.wo)
        && isfinite(trainInput.wn)
        && isfinite(trainInput.roughness)
        && isfinite(trainInput.diffuse)
        && isfinite(trainInput.specular)
        && isfinite(trainTarget.radiance)
        && getRand(1, 2, rotation.y) < NRC_BATCH_SIZE / float(params.dim.x * params.dim.y)) {
        const auto trainIdx = i % NRC_BATCH_SIZE;
        const auto inputIdx = trainIdx * NRC_INPUT_SIZE;
        const auto outputIdx = trainIdx * NRC_OUTPUT_SIZE;
        params.trainingInput[inputIdx + 0] = trainInput.position.x;
        params.trainingInput[inputIdx + 1] = trainInput.position.y;
        params.trainingInput[inputIdx + 2] = trainInput.position.z;
        params.trainingInput[inputIdx + 3] = trainInput.wo.x;
        params.trainingInput[inputIdx + 4] = trainInput.wo.y;
        params.trainingInput[inputIdx + 5] = trainInput.wn.x;
        params.trainingInput[inputIdx + 6] = trainInput.wn.y;
        params.trainingInput[inputIdx + 7] = trainInput.roughness;
        params.trainingInput[inputIdx + 8] = trainInput.diffuse.x;
        params.trainingInput[inputIdx + 9] = trainInput.diffuse.y;
        params.trainingInput[inputIdx + 10] = trainInput.diffuse.z;
        params.trainingInput[inputIdx + 11] = trainInput.specular.x;
        params.trainingInput[inputIdx + 12] = trainInput.specular.y;
        params.trainingInput[inputIdx + 13] = trainInput.specular.z;
        params.trainingTarget[outputIdx + 0] = trainTarget.radiance.x;
        params.trainingTarget[outputIdx + 1] = trainTarget.radiance.y;
        params.trainingTarget[outputIdx + 2] = trainTarget.radiance.z;
    }

    // NRC Inference
    const auto inputIdx = i * NRC_INPUT_SIZE;
    params.inferenceInput[inputIdx + 0] = nrcQuery.position.x;
    params.inferenceInput[inputIdx + 1] = nrcQuery.position.y;
    params.inferenceInput[inputIdx + 2] = nrcQuery.position.z;
    params.inferenceInput[inputIdx + 3] = nrcQuery.wo.x;
    params.inferenceInput[inputIdx + 4] = nrcQuery.wo.y;
    params.inferenceInput[inputIdx + 5] = nrcQuery.wn.x;
    params.inferenceInput[inputIdx + 6] = nrcQuery.wn.y;
    params.inferenceInput[inputIdx + 7] = nrcQuery.roughness;
    params.inferenceInput[inputIdx + 8] = nrcQuery.diffuse.x;
    params.inferenceInput[inputIdx + 9] = nrcQuery.diffuse.y;
    params.inferenceInput[inputIdx + 10] = nrcQuery.diffuse.z;
    params.inferenceInput[inputIdx + 11] = nrcQuery.specular.x;
    params.inferenceInput[inputIdx + 12] = nrcQuery.specular.y;
    params.inferenceInput[inputIdx + 13] = nrcQuery.specular.z;

    // NOTE: We should not need to prevent NaNs
    // FIXME: NaNs
    //if (isfinite(color))
    if (!(NRC_INFERENCE_FLAG & params.flags))
    params.image[i] = mix(params.image[i], make_float4(max(color, 0.0f), 1.0f), params.weight); // FIXME: Negative colors
}

extern "C" __global__ void __closesthit__ch() {
    // Get optix built-in variables
    const auto bary2 = optixGetTriangleBarycentrics();
    const auto bary = make_float3(1.0f - bary2.x - bary2.y, bary2);
    const auto data = reinterpret_cast<HitData*>(optixGetSbtDataPointer());

    // Get triangle vertices
    const auto idx = data->indexBuffer[optixGetPrimitiveIndex()];
    const auto v0 = data->vertexData[idx.x];
    const auto v1 = data->vertexData[idx.y];
    const auto v2 = data->vertexData[idx.z];

    const auto e0 = v1.position - v0.position;
    const auto e1 = v2.position - v0.position;
    const auto area = 0.5f * length(optixTransformVectorFromObjectToWorldSpace(cross(e0, e1)));

    // Interpolate normal
    const auto objectSpaceNormal = bary.x * v0.normal + bary.y * v1.normal + bary.z * v2.normal;

    // Interpolate tangent
    const auto objectSpaceTangentWithOrientation = bary.x * v0.tangent + bary.y * v1.tangent + bary.z * v2.tangent;
    const auto objectSpaceTangent = make_float3(objectSpaceTangentWithOrientation);
    const auto worldSpaceTangent = optixTransformVectorFromObjectToWorldSpace(objectSpaceTangent);

    const auto texCoord = bary.x * v0.texCoord + bary.y * v1.texCoord + bary.z * v2.texCoord;

    // Get material
    const auto material = params.materials[data->materialID];
    
    auto baseColor = material.baseColor;
    if (material.baseMap) baseColor *= make_float3(tex2D<float4>(material.baseMap, texCoord.x, texCoord.y));

    auto mr = make_float2(material.metallic, material.roughness);
    if (material.mrMap) mr *= make_float2(tex2D<float4>(material.mrMap, texCoord.x, texCoord.y));

    // NOTE: Normal mapping produces artifacts with pathtracing: See Microfacet-based Normal Mapping for Robust Monte Carlo Path Tracing by Schüssler et al. 2017 for a solution
    if (material.normalMap) { // MikkTSpace normal mapping
        const auto tangentOrientation = objectSpaceTangentWithOrientation.w;
        const auto tangentSpaceNormal = make_float3(tex2D<float4>(material.normalMap, texCoord.x, texCoord.y)) * 2.0f - 1.0f;
        const auto objectSpaceBitangent = cross(objectSpaceNormal, objectSpaceTangent) * tangentOrientation;
        setNormal(normalize(optixTransformNormalFromObjectToWorldSpace(tangentSpaceNormal.x * objectSpaceTangent + tangentSpaceNormal.y * objectSpaceBitangent + tangentSpaceNormal.z * objectSpaceNormal)));
    } else {
        setNormal(normalize(optixTransformNormalFromObjectToWorldSpace(objectSpaceNormal)));
    }
    
    setBaseColor(baseColor);
    setTangent(worldSpaceTangent);
    setMetallic(mr.x);
    setEmission(material.emission);
    setRoughness(mr.y);
    setTransmission(params.flags & TRANSMISSION_FLAG ? material.transmission : 0.0f);
    setArea(area);
    setT(optixGetRayTmax());
}

extern "C" __global__ void __miss__ms() {
    const auto dir = optixGetWorldRayDirection();
    auto sky = make_float3(0.0f);

    setEmission(sky);
    setT(INFINITY);
}