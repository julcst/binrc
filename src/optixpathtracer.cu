#include "hip/hip_runtime.h"
#include <optix_device.h>

#include "optixparams.hpp"

extern "C" __global__ void __raygen__rg() {
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();
    const uint i = idx.y * params.dim.x + idx.x;
    params.image[i] = make_uchar4(idx.x, idx.y, 0, 255);
}