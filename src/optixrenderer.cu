#include "hip/hip_runtime.h"
#include "optixrenderer.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <optix.h>
#include <optix_host.h>
#include <optix_stubs.h>
#include <optix_function_table_definition.h>
#include <optix_types.h>

#include <framework/common.hpp>

#include <iostream>
#include <array>
#include <vector>
#include <fstream>

#include <tiny-cuda-nn/common_host.h>
#include <tiny-cuda-nn/gpu_matrix.h>
#include <tiny-cuda-nn/config.h>

#include "optixir.hpp"
#include "cudautil.hpp"
#include "cudaglm.cuh"
#include "optix/params.cuh"
#include "cudamath.cuh"

std::vector<char> readBinaryFile(const std::filesystem::path& filepath) {
    std::ifstream stream{filepath, std::ios::binary};
    std::cout << "Loading " << std::filesystem::absolute(filepath) << std::endl;
    if (stream.fail()) throw std::runtime_error("Could not open file: " + std::filesystem::absolute(filepath).string());
    return {std::istreambuf_iterator<char>(stream), std::istreambuf_iterator<char>()};
}

OptixRenderer::OptixRenderer() {
    check(hipFree(nullptr)); // Initialize CUDA for this device on this thread
    check(optixInit()); // Initialize the OptiX API
    
    // Initialize the OptiX device context
    OptixDeviceContextOptions options = {
        .logCallbackFunction = [](unsigned int level, const char* tag, const char* message, void*) {
            std::cerr << "[" << tag << "] " << message << std::endl;
        },
        .logCallbackLevel = 4, // Print all log messages
        .validationMode = OPTIX_DEVICE_CONTEXT_VALIDATION_MODE_OFF,
    };
#ifdef OPTIX_DEBUG
    options.validationMode = OPTIX_DEVICE_CONTEXT_VALIDATION_MODE_ALL; // Enable all validation checks
#endif
    hipCtx_t cuCtx = nullptr; // zero means take the current context
    check(optixDeviceContextCreate(cuCtx, &options, &context));

    // Create module
    OptixModuleCompileOptions moduleCompileOptions = {
        .maxRegisterCount = OPTIX_COMPILE_DEFAULT_MAX_REGISTER_COUNT,
        .optLevel = OPTIX_COMPILE_OPTIMIZATION_DEFAULT,
        .debugLevel = OPTIX_COMPILE_DEBUG_LEVEL_MINIMAL,
        .numPayloadTypes = 0,
        .payloadTypes = nullptr,
    };
#ifdef OPTIX_DEBUG
    //moduleCompileOptions.optLevel = OPTIX_COMPILE_OPTIMIZATION_LEVEL_0; // Disable optimizations
    moduleCompileOptions.debugLevel = OPTIX_COMPILE_DEBUG_LEVEL_MODERATE; // Generate debug information
#endif
    const OptixPipelineCompileOptions pipelineCompileOptions = {
        .usesMotionBlur = false,
        .traversableGraphFlags = OPTIX_TRAVERSABLE_GRAPH_FLAG_ALLOW_SINGLE_LEVEL_INSTANCING,
        .numPayloadValues = PAYLOAD_SIZE,
        .numAttributeValues = 2,
        .exceptionFlags = OPTIX_EXCEPTION_FLAG_NONE,
        .pipelineLaunchParamsVariableName = "params",
        .usesPrimitiveTypeFlags = static_cast<unsigned int>(OPTIX_PRIMITIVE_TYPE_FLAGS_TRIANGLE),
    };

    for (size_t i = 0; i < optixir::paths.size(); i++) {
        const auto binary = readBinaryFile(optixir::paths[i]);
        check(optixModuleCreate(context, &moduleCompileOptions, &pipelineCompileOptions, binary.data(), binary.size(), nullptr, nullptr, &modules[i]));
    }

    // Create program groups
    OptixProgramGroupOptions pgOptions = {};
    const std::array programDecriptions = {
        OptixProgramGroupDesc {
            .kind = OPTIX_PROGRAM_GROUP_KIND_RAYGEN,
            .raygen = {
                .module = modules[COMBINED],
                .entryFunctionName = "__raygen__combined",
            },
        },
        OptixProgramGroupDesc {
            .kind = OPTIX_PROGRAM_GROUP_KIND_RAYGEN,
            .raygen = {
                .module = modules[REFERENCE],
                .entryFunctionName = "__raygen__reference",
            },
        },
        OptixProgramGroupDesc {
            .kind = OPTIX_PROGRAM_GROUP_KIND_MISS,
            .miss = {
                .module = modules[HIT],
                .entryFunctionName = "__miss__ms",
            },
        },
        OptixProgramGroupDesc {
            .kind = OPTIX_PROGRAM_GROUP_KIND_HITGROUP,
            .hitgroup = {
                .moduleCH = modules[HIT],
                .entryFunctionNameCH = "__closesthit__ch",
            },
        },
    };
    check(optixProgramGroupCreate(context, programDecriptions.data(), programDecriptions.size(), &pgOptions, nullptr, nullptr, programGroups.data()));

    // Create pipeline
    OptixPipelineLinkOptions pipelineLinkOptions = {
        .maxTraceDepth = MAX_BOUNCES,
    };
    check(optixPipelineCreate(context, &pipelineCompileOptions, &pipelineLinkOptions, programGroups.data(), programGroups.size(), nullptr, nullptr, &pipeline));

    // TODO: optixUtilComputeStackSizesSimplePathtracer?

    // Set up shader binding table
    std::vector<RaygenRecord> raygenRecord(2);
    check(optixSbtRecordPackHeader(programGroups[COMBINED_RG], &raygenRecord[0]));
    check(optixSbtRecordPackHeader(programGroups[REFERENCE_RG], &raygenRecord[1]));
    raygenRecords.resize_and_copy_from_host(raygenRecord);

    MissRecord missRecord;
    check(optixSbtRecordPackHeader(programGroups[MS], &missRecord));
    missRecords.resize_and_copy_from_host({missRecord});

    for (size_t i = 0; i < sbts.size(); i++) {
        sbts[i] = {
            .raygenRecord = reinterpret_cast<hipDeviceptr_t>(&raygenRecords[i]),
            .missRecordBase = reinterpret_cast<hipDeviceptr_t>(missRecords.data()),
            .missRecordStrideInBytes = sizeof(MissRecord),
            .missRecordCount = 1,
            .hitgroupRecordBase = 0,
            .hitgroupRecordStrideInBytes = sizeof(HitRecord),
            .hitgroupRecordCount = 0,
        };
    }

    params.copy_from_host({Params{}});

    nrcModel = tcnn::create_from_config(NRC_INPUT_SIZE, NRC_OUTPUT_SIZE, nlohmann::json::parse(Common::readFile("nrc.json"), nullptr, true, true));
    nrcTrainInput = tcnn::GPUMatrix<float>(NRC_INPUT_SIZE, NRC_BATCH_SIZE);
    nrcTrainOutput = tcnn::GPUMatrix<float>(NRC_OUTPUT_SIZE, NRC_BATCH_SIZE);

    std::cout << "Network: " << std::setw(2) << nrcModel.network->hyperparams()
              << "\nTrainer: " << std::setw(2) << nrcModel.trainer->hyperparams()
              << std::endl;

    getParams().trainingInput = nrcTrainInput.data();
    getParams().trainingTarget = nrcTrainOutput.data();

    nrcTrainIndex = tcnn::GPUMemory<uint>(1, true);
    nrcTrainIndex.memset(0);
    getParams().trainingIndexPtr = nrcTrainIndex.data();
}

OptixRenderer::~OptixRenderer() {
    for (auto& module : modules) check(optixModuleDestroy(module));
    check(optixPipelineDestroy(pipeline));
    check(optixDeviceContextDestroy(context));
}

void OptixRenderer::loadGLTF(const std::filesystem::path& path) {
    auto sceneData = scene.loadGLTF(context, path);
    const auto aabb = scene.getAABB();
    const auto size = aabb.max - aabb.min;

    for (auto& hitRecord : sceneData.hitRecords) optixSbtRecordPackHeader(programGroups[CH], &hitRecord);

    hitRecords.resize_and_copy_from_host(sceneData.hitRecords);
    materials.resize_and_copy_from_host(sceneData.materials);
    lightTable.resize_and_copy_from_host(sceneData.lightTable);

    for (auto& sbt : sbts) {
        sbt.hitgroupRecordBase = reinterpret_cast<hipDeviceptr_t>(hitRecords.data());
        sbt.hitgroupRecordStrideInBytes = sizeof(HitRecord);
        sbt.hitgroupRecordCount = hitRecords.size();
    }

    getParams().sceneMin = {aabb.min.x, aabb.min.y, aabb.min.z};
    getParams().sceneScale = 1.0f / std::max(size.x, std::max(size.y, size.z));
    getParams().materials = materials.data();
    getParams().lightTable = lightTable.data();
    getParams().lightTableSize = lightTable.size();
    getParams().handle = sceneData.handle;

    std::cout << "Min: (" << getParams().sceneMin.x << ", " << getParams().sceneMin.y << ", " << getParams().sceneMin.z << ") Scale: " << getParams().sceneScale << std::endl;

    reset();
    lossHistory.clear();
}

void OptixRenderer::setCamera(const mat4& clipToWorld) {
    getParams().clipToWorld = glmToCuda(clipToWorld);
}

__global__ void visualizeInference(Params* params) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= params->dim.x || y >= params->dim.y) return;
    const int i = y * params->dim.x + x;
    const int idxIn = i * NRC_INPUT_SIZE;
    const int idxOut = i * NRC_OUTPUT_SIZE;
    auto inference = make_float3(params->inferenceOutput[idxOut + 0], params->inferenceOutput[idxOut + 1], params->inferenceOutput[idxOut + 2]);
    if (!isfinite(inference)) return;
    if (params->inferenceMode == InferenceMode::RAW_CACHE) {
        params->image[i] = make_float4(inference, 1.0f);
    } else {
        const auto diffuse = make_float3(params->inferenceInput[idxIn + 8], params->inferenceInput[idxIn + 9], params->inferenceInput[idxIn + 10]);
        const auto specular = make_float3(params->inferenceInput[idxIn + 11], params->inferenceInput[idxIn + 12], params->inferenceInput[idxIn + 13]);
        const auto throughput = params->inferenceThroughput[i];
        params->image[i] += params->weight * make_float4(inference * (diffuse + specular) * throughput, 1.0f);
    }
}

void OptixRenderer::train() {
    // TODO: Permute the training data
    for (uint32_t offset = 0; offset < NRC_BATCH_SIZE; offset += NRC_SUBBATCH_SIZE) {
        auto ctx = nrcModel.trainer->training_step(nrcTrainInput.slice_cols(offset, NRC_SUBBATCH_SIZE), nrcTrainOutput.slice_cols(offset, NRC_SUBBATCH_SIZE));
        float loss = nrcModel.trainer->loss(*ctx);
        lossHistory.push_back(loss);
    }
}

void OptixRenderer::render(vec4* image, uvec2 dim) {
    getParams().image = reinterpret_cast<float4*>(image);
    getParams().dim = make_uint2(dim.x, dim.y);
    const auto prevTrainIndex = nrcTrainIndex.at(0);
    
    ensureSobol(getParams().sample);
    check(optixLaunch(pipeline, nullptr, reinterpret_cast<hipDeviceptr_t>(params.data()), sizeof(Params), &sbts[0], dim.x, dim.y, 1));
    check(hipDeviceSynchronize()); // Wait for the renderer to finish

    if (!scene.isEmpty()) train();

    if (getParams().inferenceMode != InferenceMode::NO_INFERENCE) {
        nrcModel.network->inference(nrcInferenceInput, nrcInferenceOutput);

        dim3 block(16, 16);
        dim3 grid((dim.x + block.x - 1) / block.x, (dim.y + block.y - 1) / block.y);
        visualizeInference<<<grid, block>>>(params.data());
        check(hipDeviceSynchronize()); // Wait for the visualization to finish
    }
    
    getParams().sample++;
    getParams().weight = 1.0f / static_cast<float>(getParams().sample);
    //std::cout << nrcTrainIndex.at(0) - prevTrainIndex << std::endl;
}

void OptixRenderer::generateSobol(uint offset, uint n) {
    randSequence.resize(n * RAND_SEQUENCE_DIMS);

    getParams().sequenceStride = n;
    getParams().sequenceOffset = offset;
    getParams().randSequence = randSequence.data();

    // NOTE: We rebuild the generator, this makes regeneration slow but saves memory
    hiprandGenerator_t generator;
    check(hiprandCreateGenerator(&generator, HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL32));
    check(hiprandSetQuasiRandomGeneratorDimensions(generator, RAND_SEQUENCE_DIMS)); // 4 dimensions for 4D Sobol sequence
    check(hiprandSetGeneratorOffset(generator, offset)); // Reset the sequence
    check(hiprandGenerateUniform(generator, randSequence.data(), randSequence.size()));
    check(hipDeviceSynchronize()); // Wait for the generator to finish
    check(hiprandDestroyGenerator(generator));
}

void OptixRenderer::ensureSobol(uint sample) {
    if (sample < getParams().sequenceOffset || sample >= getParams().sequenceOffset + getParams().sequenceStride) {
        // std::cout << std::format("Regenerating Sobol sequence for samples [{},{})", sample, sample + RAND_SEQUENCE_CACHE_SIZE) << std::endl;
        // C++17
        std::cout << "Regenerating Sobol sequence for samples [" << sample << "," << sample + RAND_SEQUENCE_CACHE_SIZE << ")" << std::endl;
        generateSobol(sample, RAND_SEQUENCE_CACHE_SIZE);
    }
}

void OptixRenderer::resize(uvec2 dim) {
    // Generate inference input and output buffers
    auto inferenceBatchSize = dim.x * dim.y;
    inferenceBatchSize += tcnn::BATCH_SIZE_GRANULARITY - inferenceBatchSize % tcnn::BATCH_SIZE_GRANULARITY; // Round up to the next multiple of BATCH_SIZE_GRANULARITY

    nrcInferenceInput = tcnn::GPUMatrix<float>(NRC_INPUT_SIZE, inferenceBatchSize);
    nrcInferenceOutput = tcnn::GPUMatrix<float>(NRC_OUTPUT_SIZE, inferenceBatchSize);
    nrcInferenceThroughput = tcnn::GPUMemory<float3>(inferenceBatchSize);

    // Generate the Cranley-Patterson-Rotation per pixel
    // NOTE: We rebuild the generator on resize, this makes resize slow but saves memory
    rotationTable.resize(dim.x * dim.y * ROTATIONS_PER_PIXEL);

    hiprandGenerator_t generator;
    check(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_XORWOW));
    check(hiprandGenerateUniform(generator, rotationTable.data(), rotationTable.size()));
    check(hiprandDestroyGenerator(generator));

    getParams().inferenceInput = nrcInferenceInput.data();
    getParams().inferenceOutput = nrcInferenceOutput.data();
    getParams().inferenceThroughput = nrcInferenceThroughput.data();
    getParams().rotationTable = rotationTable.data();

    check(hipDeviceSynchronize()); // Wait for the generator to finish
}

void OptixRenderer::reset() {
    getParams().sample = 0;
    getParams().weight = 1.0f;
}

void OptixRenderer::resetNRC() {
    nrcModel.trainer->initialize_params();
    lossHistory.clear();
}