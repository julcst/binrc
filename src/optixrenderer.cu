#include "hip/hip_runtime.h"
#include "optixrenderer.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <optix.h>
#include <optix_host.h>
#include <optix_stubs.h>
#include <optix_function_table_definition.h>
#include <optix_types.h>

#include <framework/common.hpp>

#include <iostream>
#include <array>
#include <vector>
#include <fstream>

#include <tiny-cuda-nn/common_host.h>
#include <tiny-cuda-nn/gpu_matrix.h>
#include <tiny-cuda-nn/config.h>

#include "optixir.hpp"
#include "cudautil.hpp"
#include "cudaglm.cuh"
#include "optix/params.cuh"
#include "cudamath.cuh"
#include "optix/sampling.cuh"

OptixRenderer::OptixRenderer() {
    check(hipFree(nullptr)); // Initialize CUDA for this device on this thread
    check(optixInit()); // Initialize the OptiX API
    
    // Initialize the OptiX device context
    OptixDeviceContextOptions options = {
        .logCallbackFunction = [](unsigned int level, const char* tag, const char* message, void*) {
            std::cerr << "[" << tag << "] " << message << std::endl;
        },
        .logCallbackLevel = 4, // Print all log messages
        .validationMode = OPTIX_DEVICE_CONTEXT_VALIDATION_MODE_OFF,
    };
#ifdef OPTIX_DEBUG
    options.validationMode = OPTIX_DEVICE_CONTEXT_VALIDATION_MODE_ALL; // Enable all validation checks
#endif
    hipCtx_t cuCtx = nullptr; // zero means take the current context
    check(optixDeviceContextCreate(cuCtx, &options, &context));

    // Create module
    OptixModuleCompileOptions moduleCompileOptions = {
        .maxRegisterCount = OPTIX_COMPILE_DEFAULT_MAX_REGISTER_COUNT,
        .optLevel = OPTIX_COMPILE_OPTIMIZATION_DEFAULT,
        .debugLevel = OPTIX_COMPILE_DEBUG_LEVEL_MINIMAL,
        .numPayloadTypes = 0,
        .payloadTypes = nullptr,
    };
#ifdef OPTIX_DEBUG
    //moduleCompileOptions.optLevel = OPTIX_COMPILE_OPTIMIZATION_LEVEL_0; // Disable optimizations
    moduleCompileOptions.debugLevel = OPTIX_COMPILE_DEBUG_LEVEL_MODERATE; // Generate debug information
#endif
    const OptixPipelineCompileOptions pipelineCompileOptions = {
        .usesMotionBlur = false,
        .traversableGraphFlags = OPTIX_TRAVERSABLE_GRAPH_FLAG_ALLOW_SINGLE_LEVEL_INSTANCING,
        .numPayloadValues = PAYLOAD_SIZE,
        .numAttributeValues = 2,
        .exceptionFlags = OPTIX_EXCEPTION_FLAG_NONE,
        .pipelineLaunchParamsVariableName = "params",
        .usesPrimitiveTypeFlags = static_cast<unsigned int>(OPTIX_PRIMITIVE_TYPE_FLAGS_TRIANGLE),
    };

    for (size_t i = 0; i < optixir::paths.size(); i++) {
        const auto binary = Common::readBinaryFile(optixir::paths[i]);
        check(optixModuleCreate(context, &moduleCompileOptions, &pipelineCompileOptions, binary.data(), binary.size(), nullptr, nullptr, &modules[i]));
    }

    // Create program groups
    OptixProgramGroupOptions pgOptions = {};
    const std::array programDecriptions = {
        OptixProgramGroupDesc {
            .kind = OPTIX_PROGRAM_GROUP_KIND_RAYGEN,
            .raygen = {
                .module = modules[optixir::COMBINED],
                .entryFunctionName = "__raygen__combined",
            },
        },
        OptixProgramGroupDesc {
            .kind = OPTIX_PROGRAM_GROUP_KIND_RAYGEN,
            .raygen = {
                .module = modules[optixir::REFERENCE],
                .entryFunctionName = "__raygen__reference",
            },
        },
        OptixProgramGroupDesc {
            .kind = OPTIX_PROGRAM_GROUP_KIND_RAYGEN,
            .raygen = {
                .module = modules[optixir::TRAIN_FORWARD],
                .entryFunctionName = "__raygen__",
            },
        },
        OptixProgramGroupDesc {
            .kind = OPTIX_PROGRAM_GROUP_KIND_RAYGEN,
            .raygen = {
                .module = modules[optixir::TRAIN_BACKWARD],
                .entryFunctionName = "__raygen__",
            },
        },
        OptixProgramGroupDesc {
            .kind = OPTIX_PROGRAM_GROUP_KIND_RAYGEN,
            .raygen = {
                .module = modules[optixir::INFERENCE],
                .entryFunctionName = "__raygen__",
            },
        },
        OptixProgramGroupDesc {
            .kind = OPTIX_PROGRAM_GROUP_KIND_MISS,
            .miss = {
                .module = modules[optixir::HIT],
                .entryFunctionName = "__miss__ms",
            },
        },
        OptixProgramGroupDesc {
            .kind = OPTIX_PROGRAM_GROUP_KIND_HITGROUP,
            .hitgroup = {
                .moduleCH = modules[optixir::HIT],
                .entryFunctionNameCH = "__closesthit__ch",
            },
        },
    };
    check(optixProgramGroupCreate(context, programDecriptions.data(), programDecriptions.size(), &pgOptions, nullptr, nullptr, programGroups.data()));

    // Create pipeline
    OptixPipelineLinkOptions pipelineLinkOptions = {
        .maxTraceDepth = MAX_BOUNCES,
    };
    check(optixPipelineCreate(context, &pipelineCompileOptions, &pipelineLinkOptions, programGroups.data(), programGroups.size(), nullptr, nullptr, &pipeline));

    // TODO: optixUtilComputeStackSizesSimplePathtracer?

    // Set up shader binding table
    std::vector<RaygenRecord> raygenRecord(sbts.size());
    check(optixSbtRecordPackHeader(programGroups[COMBINED], &raygenRecord[COMBINED]));
    check(optixSbtRecordPackHeader(programGroups[REFERENCE], &raygenRecord[REFERENCE]));
    check(optixSbtRecordPackHeader(programGroups[TRAIN_FORWARD], &raygenRecord[TRAIN_FORWARD]));
    check(optixSbtRecordPackHeader(programGroups[TRAIN_BACKWARD], &raygenRecord[TRAIN_BACKWARD]));
    check(optixSbtRecordPackHeader(programGroups[INFERENCE], &raygenRecord[INFERENCE]));
    raygenRecords.resize_and_copy_from_host(raygenRecord);

    MissRecord missRecord;
    check(optixSbtRecordPackHeader(programGroups[MISS], &missRecord));
    missRecords.resize_and_copy_from_host({missRecord});

    for (size_t i = 0; i < sbts.size(); i++) {
        sbts[i] = {
            .raygenRecord = reinterpret_cast<hipDeviceptr_t>(&raygenRecords[i]),
            .missRecordBase = reinterpret_cast<hipDeviceptr_t>(missRecords.data()),
            .missRecordStrideInBytes = sizeof(MissRecord),
            .missRecordCount = 1,
            .hitgroupRecordBase = 0,
            .hitgroupRecordStrideInBytes = sizeof(HitRecord),
            .hitgroupRecordCount = 0,
        };
    }

    nrcModel = tcnn::create_from_config(NRC_INPUT_SIZE, NRC_OUTPUT_SIZE, nlohmann::json::parse(Common::readFile("nrc.json"), nullptr, true, true));
    nrcTrainInput = tcnn::GPUMatrix<float>(NRC_INPUT_SIZE, NRC_BATCH_SIZE);
    nrcTrainOutput = tcnn::GPUMatrix<float>(NRC_OUTPUT_SIZE, NRC_BATCH_SIZE);

    std::cout << "Network: " << std::setw(2) << nrcModel.network->hyperparams()
              << "\nTrainer: " << std::setw(2) << nrcModel.trainer->hyperparams()
              << std::endl;
    
    params.trainingInput = nrcTrainInput.data();
    params.trainingTarget = nrcTrainOutput.data();
    params.selfLearningBounces = selfLearningBounces.data();
    params.selfLearningQueries = selfLearningQueries.data();

    nrcTrainIndex.memset(0);
    params.trainingIndexPtr = nrcTrainIndex.data();

    nrcLightSamples.memset(0);
    params.lightSamples = nrcLightSamples.data();

    params.brdfLUT = brdfLUT.texObj;
}

OptixRenderer::~OptixRenderer() {
    for (auto& module : modules) check(optixModuleDestroy(module));
    check(optixPipelineDestroy(pipeline));
    check(optixDeviceContextDestroy(context));
}

__global__ void testSceneSampling(const uint sampleCount, const Instance* instances, const uint instanceCount, const Material* materials) {
    const uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= sampleCount) return;

    hiprandStatePhilox4_32_10_t state;
    hiprand_init(0, i, 0, &state);
    const auto rand = hiprand_uniform4(&state);

    const auto surf = sampleScene(instances, instanceCount, materials, rand.x, make_float2(rand.z, rand.w));
    printf("Sample %d: %f %f %f %f %f %f %f %f %f\n", i, surf.position.x, surf.position.y, surf.position.z, surf.normal.x, surf.normal.y, surf.normal.z, surf.baseColor.x, surf.baseColor.y, surf.baseColor.z);
}

void OptixRenderer::loadGLTF(const std::filesystem::path& path) {
    auto sceneData = scene.loadGLTF(context, path);
    const auto aabb = scene.getAABB();
    const auto size = aabb.max - aabb.min;

    for (auto& hitRecord : sceneData.hitRecords) optixSbtRecordPackHeader(programGroups[CLOSEST_HIT], &hitRecord);

    hitRecords.resize_and_copy_from_host(sceneData.hitRecords);
    materials.resize_and_copy_from_host(sceneData.materials);
    lightTable.resize_and_copy_from_host(sceneData.lightTable);
    instances.resize_and_copy_from_host(sceneData.instances);

    for (auto& sbt : sbts) {
        sbt.hitgroupRecordBase = reinterpret_cast<hipDeviceptr_t>(hitRecords.data());
        sbt.hitgroupRecordStrideInBytes = sizeof(HitRecord);
        sbt.hitgroupRecordCount = hitRecords.size();
    }

    params.sceneMin = {aabb.min.x, aabb.min.y, aabb.min.z};
    params.sceneScale = 1.0f / std::max(size.x, std::max(size.y, size.z));
    params.materials = materials.data();
    params.lightTable = lightTable.data();
    params.lightTableSize = lightTable.size();
    params.handle = sceneData.handle;

    std::cout << "Min: (" << params.sceneMin.x << ", " << params.sceneMin.y << ", " << params.sceneMin.z << ") Scale: " << params.sceneScale << std::endl;

    // Test scene sampling
    const uint sampleCount = 100;
    const uint blockSize = 256;
    const uint blockCount = (sampleCount + blockSize - 1) / blockSize;
    testSceneSampling<<<blockCount, blockSize>>>(sampleCount, instances.data(), instances.size(), materials.data());

    reset();
    lossHistory.clear();
}

void OptixRenderer::setCamera(const mat4& clipToWorld) {
    params.clipToWorld = glmToCuda(clipToWorld);
}

__global__ void visualizeInference(Params* params) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= params->dim.x || y >= params->dim.y) return;

    const int i = y * params->dim.x + x;
    const int idxIn = i * NRC_INPUT_SIZE;
    const int idxOut = i * NRC_OUTPUT_SIZE;
    auto inference = make_float3(params->inferenceOutput[idxOut + 0], params->inferenceOutput[idxOut + 1], params->inferenceOutput[idxOut + 2]);

    const auto throughput = params->inferenceThroughput[i];

    if (throughput.x <= 0.0f && throughput.y <= 0.0f && throughput.z <= 0.0f) return;

    if (params->inferenceMode == InferenceMode::RAW_CACHE) {
        params->image[i] = make_float4(inference, 1.0f);
    } else {
        const auto diffuse = make_float3(params->inferenceInput[idxIn + 8], params->inferenceInput[idxIn + 9], params->inferenceInput[idxIn + 10]);
        const auto specular = make_float3(params->inferenceInput[idxIn + 11], params->inferenceInput[idxIn + 12], params->inferenceInput[idxIn + 13]);
        params->image[i] += params->weight * make_float4(inference * (diffuse + specular) * throughput, 1.0f);
        // params->image[i] = make_float4(throughput, 1.0f);
    }
}

__device__ inline void writeNRCInput(float* dest, uint idx, const NRCInput& input) {
    auto to = dest + idx * NRC_INPUT_SIZE;
    to[0] = input.position.x;
    to[1] = input.position.y;
    to[2] = input.position.z;
    to[3] = input.wo.x;
    to[4] = input.wo.y;
    to[5] = input.wn.x;
    to[6] = input.wn.y;
    to[7] = input.roughness;
    to[8] = input.diffuse.x;
    to[9] = input.diffuse.y;
    to[10] = input.diffuse.z;
    to[11] = input.specular.x;
    to[12] = input.specular.y;
    to[13] = input.specular.z;
}

__device__ inline void writeNRCOutput(float* dest, uint idx, const float3& radiance) {
    auto to = dest + idx * NRC_OUTPUT_SIZE;
    to[0] = radiance.x;
    to[1] = radiance.y;
    to[2] = radiance.z;
}

__device__ inline NRCInput encodeInput(const Params* params, const float3& wo, const Surface& surf, const hipTextureObject_t brdfLUT) {
    const auto F0 = mix(make_float3(0.04f), surf.baseColor, surf.metallic);
    const auto lut = tex2D<float4>(brdfLUT, surf.roughness, dot(surf.normal, wo));
    const auto specular = F0 * lut.x + lut.y;
    const auto albedo = (1.0f - surf.metallic) * surf.baseColor;
    return {
        .position = params->sceneScale * (surf.position - params->sceneMin),
        .wo = toNormSpherical(wo),
        .wn = toNormSpherical(surf.normal),
        .roughness = surf.roughness * surf.roughness,
        .diffuse = albedo,
        .specular = specular,
    };
}

__global__ void generateDummySamples(const uint sampleCount, Params* params, const Instance* instances, const uint instanceCount, const Material* materials) {
    const uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= sampleCount) return;

    hiprandStatePhilox4_32_10_t state;
    hiprand_init(0, i, params->trainingRound * 5, &state);
    const auto rand = hiprand_uniform4(&state);

    const auto surf = sampleScene(instances, instanceCount, materials, rand.x, {rand.y, rand.z});
    const auto wo = buildTBN(surf.normal) * sampleCosineHemisphere({rand.w, hiprand_uniform(&state)});

    const auto input = encodeInput(params, wo, surf, params->brdfLUT);
    const auto idx = atomicAdd(params->trainingIndexPtr, 1u) % NRC_BATCH_SIZE;
    writeNRCInput(params->trainingInput, idx, input);
    writeNRCOutput(params->trainingTarget, idx, make_float3(0.0f));
}

__global__ void applySelfLearning(unsigned int numQueries, std::array<TrainBounce, TRAIN_DEPTH>* selfLearningBounces, float* nrcQueries, float* nrcOutput, float* trainTarget) {
    const auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numQueries) return;

    const int idxIn = i * NRC_INPUT_SIZE;
    const int idxOut = i * NRC_OUTPUT_SIZE;

    const auto bounces = selfLearningBounces[i];
    auto inference = make_float3(nrcOutput[idxOut + 0], nrcOutput[idxOut + 1], nrcOutput[idxOut + 2]);
    const auto diffuse = make_float3(nrcQueries[idxIn + 8], nrcQueries[idxIn + 9], nrcQueries[idxIn + 10]);
    const auto specular = make_float3(nrcQueries[idxIn + 11], nrcQueries[idxIn + 12], nrcQueries[idxIn + 13]);
    inference *= (diffuse + specular);

    for (const auto bounce : bounces) {
        if (!bounce.isValid) continue;
        /*printf("Bounce %d: (%f %f %f) * Inference (%f %f %f) + Radiance (%f %f %f)\n", bounce.index, bounce.throughput.x, bounce.throughput.y, bounce.throughput.z, 
               inference.x, inference.y, inference.z, bounce.radiance.x, bounce.radiance.y, bounce.radiance.z);*/
        writeNRCOutput(trainTarget, bounce.index, bounce.reflectanceFactorizationTerm * (inference * bounce.throughput + bounce.radiance));
    }
}

// TODO: Could do multiple smaller training steps per frame
void OptixRenderer::train() {
    nrcLightSamples.memset(0);
    check(hipDeviceSynchronize()); // Wait for the renderer to finish

    // Generate training samples
    const float balanceRatio = 1.0f / params.balanceWeight;
    const auto totalTrainingSamples = NRC_BATCH_SIZE / TRAIN_DEPTH;
    const uint forwardSamples = totalTrainingSamples * (1.0f - trainingDirection);
    const uint backwardSamples = (totalTrainingSamples - forwardSamples) * balanceRatio;
    if (forwardSamples) check(optixLaunch(pipeline, nullptr, reinterpret_cast<hipDeviceptr_t>(paramsBuffer.data()), sizeof(Params), &sbts[TRAIN_FORWARD], forwardSamples, 1, 1));
    if (backwardSamples) {
        check(optixLaunch(pipeline, nullptr, reinterpret_cast<hipDeviceptr_t>(paramsBuffer.data()), sizeof(Params), &sbts[TRAIN_BACKWARD], backwardSamples, 1, 1));
    }
    check(hipDeviceSynchronize()); // Wait for the renderer to finish
    
    uint nL = 0; // Real number of light samples
    nrcLightSamples.copy_to_host(&nL);
    // (nL + nD) * r = nL    and    r = 1 / w    =>    nD = nL * w - nL
    uint nD = nL * params.balanceWeight - nL;
    //std::cout << "nL: " << nL << " nD: " << nD << std::endl;
    // FIXME: Upsides too dark
    if (nD) generateDummySamples<<<(nD + 255) / 256, 256>>>(nD, paramsBuffer.data(), instances.data(), instances.size(), materials.data());
    check(hipDeviceSynchronize()); // Wait for the renderer to finish

    if (params.flags & SELF_LEARNING_FLAG) {
        nrcModel.network->inference(selfLearningQueries, selfLearningInference, false); // Do not apply EMA here
        const auto block = 256;
        const auto grid = (forwardSamples + block - 1) / block;
        applySelfLearning<<<grid, block>>>(forwardSamples, selfLearningBounces.data(), selfLearningQueries.data(), selfLearningInference.data(), nrcTrainOutput.data());
    }
    
    params.trainingRound++;

    // Perform training steps
    for (uint32_t offset = 0; offset < NRC_BATCH_SIZE; offset += NRC_SUBBATCH_SIZE) {
        // TODO: Use pdf
        // TODO: Limit training to the samples generated in this step to improve performance
        auto ctx = nrcModel.trainer->training_step(nrcTrainInput.slice_cols(offset, NRC_SUBBATCH_SIZE), nrcTrainOutput.slice_cols(offset, NRC_SUBBATCH_SIZE));
        float loss = nrcModel.trainer->loss(*ctx);
        lossHistory.push_back(loss);
    }
}

void OptixRenderer::render(vec4* image, uvec2 dim) {

    // Update parameters
    params.image = reinterpret_cast<float4*>(image);
    params.dim = make_uint2(dim.x, dim.y);
    ensureSobol(params.sample);
    
    // Copy host parameters to device
    paramsBuffer.copy_from_host(&params, 1);
    check(hipDeviceSynchronize()); // Wait for the copy to finish

    if (!scene.isEmpty() && enableTraining) train();

    if (params.inferenceMode == InferenceMode::NO_INFERENCE) { // Reference
        check(optixLaunch(pipeline, nullptr, reinterpret_cast<hipDeviceptr_t>(paramsBuffer.data()), sizeof(Params), &sbts[REFERENCE], dim.x, dim.y, 1));
        check(hipDeviceSynchronize()); // Wait for the renderer to finish
    } else {
        check(optixLaunch(pipeline, nullptr, reinterpret_cast<hipDeviceptr_t>(paramsBuffer.data()), sizeof(Params), &sbts[INFERENCE], dim.x, dim.y, 1));
        check(hipDeviceSynchronize()); // Wait for the renderer to finish

        nrcModel.network->inference(nrcInferenceInput, nrcInferenceOutput);

        dim3 block(16, 16);
        dim3 grid((dim.x + block.x - 1) / block.x, (dim.y + block.y - 1) / block.y);
        visualizeInference<<<grid, block>>>(paramsBuffer.data());
        check(hipDeviceSynchronize()); // Wait for the visualization to finish
    }
    
    params.sample++;
    params.weight = 1.0f / static_cast<float>(params.sample);
}

void OptixRenderer::generateSobol(uint offset, uint n) {
    randSequence.resize(n * RAND_SEQUENCE_DIMS);

    params.sequenceStride = n;
    params.sequenceOffset = offset;
    params.randSequence = randSequence.data();

    // NOTE: We rebuild the generator, this makes regeneration slow but saves memory
    hiprandGenerator_t generator;
    check(hiprandCreateGenerator(&generator, HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL32));
    check(hiprandSetQuasiRandomGeneratorDimensions(generator, RAND_SEQUENCE_DIMS)); // 4 dimensions for 4D Sobol sequence
    check(hiprandSetGeneratorOffset(generator, offset)); // Reset the sequence
    check(hiprandGenerateUniform(generator, randSequence.data(), randSequence.size()));
    check(hipDeviceSynchronize()); // Wait for the generator to finish
    check(hiprandDestroyGenerator(generator));
}

void OptixRenderer::ensureSobol(uint sample) {
    if (sample < params.sequenceOffset || sample >= params.sequenceOffset + params.sequenceStride) {
        // std::cout << std::format("Regenerating Sobol sequence for samples [{},{})", sample, sample + RAND_SEQUENCE_CACHE_SIZE) << std::endl;
        // C++17
        std::cout << "Regenerating Sobol sequence for samples [" << sample << "," << sample + RAND_SEQUENCE_CACHE_SIZE << ")" << std::endl;
        generateSobol(sample, RAND_SEQUENCE_CACHE_SIZE);
    }
}

void OptixRenderer::resize(uvec2 dim) {
    // Generate inference input and output buffers
    auto inferenceBatchSize = dim.x * dim.y;
    inferenceBatchSize += tcnn::BATCH_SIZE_GRANULARITY - inferenceBatchSize % tcnn::BATCH_SIZE_GRANULARITY; // Round up to the next multiple of BATCH_SIZE_GRANULARITY

    nrcInferenceInput = tcnn::GPUMatrix<float>(NRC_INPUT_SIZE, inferenceBatchSize);
    nrcInferenceOutput = tcnn::GPUMatrix<float>(NRC_OUTPUT_SIZE, inferenceBatchSize);
    nrcInferenceThroughput = tcnn::GPUMemory<float3>(inferenceBatchSize);

    // Generate the Cranley-Patterson-Rotation per pixel
    // NOTE: We rebuild the generator on resize, this makes resize slow but saves memory
    rotationTable.resize(dim.x * dim.y);

    hiprandGenerator_t generator;
    check(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_XORWOW));
    check(hiprandGenerateUniform(generator, reinterpret_cast<float*>(rotationTable.data()), rotationTable.size() * 2));
    check(hiprandDestroyGenerator(generator));

    params.inferenceInput = nrcInferenceInput.data();
    params.inferenceOutput = nrcInferenceOutput.data();
    params.inferenceThroughput = nrcInferenceThroughput.data();
    params.rotationTable = rotationTable.data();

    check(hipDeviceSynchronize()); // Wait for the generator to finish
}

void OptixRenderer::reset() {
    params.sample = 0;
    params.weight = 1.0f;
}

void OptixRenderer::resetNRC() {
    nrcModel.trainer->initialize_params();
    lossHistory.clear();
}