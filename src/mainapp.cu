#include "mainapp.cuh"

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <optix_types.h>

#include <framework/gl/buffer.hpp>
#include <framework/imguiutil.hpp>

#include <glm/glm.hpp>
#include <glm/gtc/type_ptr.hpp>
using namespace glm;

#include <imgui.h>
#include <misc/cpp/imgui_stdlib.h>

#include <iostream>
#include <format>
#include <filesystem>

#include "cudautil.hpp"

const std::string vs = R"(#version 460 core
layout(location = 0) in vec2 position;
out vec2 texCoord;
void main() {
    gl_Position = vec4(position, 0.0, 1.0);
    texCoord = position * 0.5 + 0.5;
})";

const std::string fs = R"(#version 460 core
in vec2 texCoord;
out vec4 fragColor;
layout(location = 0) uniform sampler2D tex;
layout(location = 1) uniform float exposure;
void main() {
    fragColor = texture(tex, texCoord) * exposure;
})";

std::vector<std::filesystem::path> scanFolder(const std::filesystem::path& folder) {
    std::vector<std::filesystem::path> files;
    try {
        for (auto& f : std::filesystem::directory_iterator(folder)) {
            if (f.path().extension() == ".glb"|| f.path().extension() == ".gltf") files.push_back(f.path());
        } 
        return files;
    } catch (const std::filesystem::filesystem_error& e) {
        std::cerr << e.what() << std::endl;
        return files;
    }
}

std::filesystem::path getNumberedPath(const std::filesystem::path& basePath, const std::string& extension) {
    int number = 0;
    std::filesystem::path path;
    do {
        // C++20
        path = basePath / (std::format("screenshot_{:03d}.{}", number, extension));
        // C++17
        // std::stringstream ss;
        // ss << basePath.string() << std::setw(3) << std::setfill('0') << number << "." << extension;
        // path = ss.str();
        number++;
    } while (std::filesystem::exists(path));
    return path;
}

MainApp::MainApp() : App(800, 800) {
    printCudaDevices();

    fullscreenTriangle.load(Mesh::FULLSCREEN_VERTICES, Mesh::FULLSCREEN_INDICES);

    blitProgram.loadSource(vs, fs);
    blitProgram.use();
    blitProgram.set(1, exposure);

    camera.worldPosition = vec3(0.0f, 0.0f, -2.0f);
    camera.invalidate();

    folder = std::filesystem::current_path().parent_path().string();
    scenes = scanFolder(folder);

    setVSync(false);
}

MainApp::~MainApp() {
    check(hipGraphicsUnregisterResource(cudaPboResource));
}

void MainApp::resize(const ivec2& res) {
    bufferDim = uvec2(res);
    if (cudaPboResource) check(hipGraphicsUnregisterResource(cudaPboResource)); // Unregister the old resource to prevent memory leak
    pbo.allocate(res.x * res.y * sizeof(vec4), GL_STREAM_DRAW);
    check(hipGraphicsGLRegisterBuffer(&cudaPboResource, pbo.handle, cudaGraphicsMapFlagsWriteDiscard));
    blitTexture = Texture<GL_TEXTURE_2D>();
    blitTexture.allocate2D(GL_RGBA32F, res.x, res.y);
    blitTexture.bindTextureUnit(0);
    camera.resize(float(res.x) / float(res.y));
    renderer.reset();
    renderer.resize(uvec2(res));
}

void MainApp::resizeCallback(const vec2& res) { 
    resize(res); // TODO: Fix scaling issues
}

void MainApp::keyCallback(Key key, Action action, Modifier modifier) {
    if (action != Action::PRESS) return;
    switch (key) {
        case Key::ESC:
            close();
            break;
        case Key::T:
            imguiEnabled = !imguiEnabled;
            break;
        case Key::C:
            takeScreenshot(getNumberedPath("screenshot_", "png").string());
            break;
        case Key::X:
            blitTexture.writeToFile(getNumberedPath("screenshot_", "hdr").string());
            break;
        case Key::J:
            std::cout << renderer.getConfig().dump(4) << std::endl;
            break;
        case Key::F:
            camera.target = vec3(0.0f, 0.0f, 0.0f);
            camera.invalidate();
            break;
        default:
            break;
    }
}

void MainApp::scrollCallback(float xamount, float yamount) {
    camera.zoom(yamount);
}

void MainApp::moveCallback(const vec2& movement, bool leftButton, bool rightButton, bool middleButton) {
    if (leftButton) camera.orbit(movement * 0.01f);
}

void MainApp::buildImGui() {
    ImGui::StatisticsWindow(delta, resolution);

    ImGui::Begin("Settings", nullptr, ImGuiWindowFlags_AlwaysAutoResize);
    bool reset = false;

    if (ImGui::CollapsingHeader("Scene", ImGuiTreeNodeFlags_DefaultOpen)) {
        std::string folder_str = folder.string();
        if (ImGui::InputText("Folder", &folder_str)) {
            folder = folder_str;
            scenes = scanFolder(folder);
            sceneID = 0;
        }
        if (ImGui::FileCombo("Scene##2", &sceneID, scenes)) {
            renderer.loadGLTF(scenes.at(sceneID));
        }
        ImGui::SliderFloat("Scene Epsilon", &renderer.params.sceneEpsilon, 1e-6f, 1e-1f, "%f", ImGuiSliderFlags_Logarithmic);
        for (size_t i = 0; i < renderer.scene.cameras.size(); i++) {
            if (ImGui::Button(renderer.scene.cameras[i].first.c_str())) {
                auto scale = mat4(1.0f);
                scale[0][0] = camera.aspectRatio;
                const auto clipToWorld = renderer.scene.cameras[i].second * scale;
                renderer.setCamera(clipToWorld);
                reset = true;
            }
            if (i < renderer.scene.cameras.size() - 1) ImGui::SameLine();
        }
    }

    if (ImGui::CollapsingHeader("Rendering", ImGuiTreeNodeFlags_DefaultOpen)) {
        ImGui::Text("Sample: %d", renderer.params.sample);
        if (ImGui::SliderFloat("Exposure", &exposure, 0.1f, 10.0f, "%.1f", ImGuiSliderFlags_Logarithmic)) blitProgram.set(1, exposure);
        reset |= ImGui::EnumCombo("Inference Mode", &renderer.params.inferenceMode, INFERENCE_MODES);
        reset |= ImGui::FlagCheckbox("Enable Light Tracing Fix", &renderer.params.flags, LIGHT_TRACE_FIX_FLAG);
        reset |= ImGui::SliderFloat("Variance Tradeoff", &renderer.params.varianceTradeoff, 0.0f, 1.0f, "%.3f");

        switch (renderer.params.inferenceMode) {
            case InferenceMode::NO_INFERENCE:
                ImGui::SliderFloat("Russian Roulette", &renderer.params.russianRouletteWeight, 1.0f, 10.0f, "%.1f");
                reset |= ImGui::FlagCheckbox("Enable NEE", &renderer.params.flags, NEE_FLAG);
                reset |= ImGui::FlagCheckbox("Enable Transmission", &renderer.params.flags, TRANSMISSION_FLAG);
                break;
        }
        reset |= ImGui::SliderInt("Max Path Length", reinterpret_cast<int*>(&renderer.params.maxPathLength), 1, MAX_BOUNCES);
    }

    if (ImGui::CollapsingHeader("NRC", ImGuiTreeNodeFlags_DefaultOpen)) {
        ImGui::Checkbox("Enable Training", &renderer.enableTraining);
        ImGui::FlagCheckbox("Enable Diffuse Encoding", &renderer.params.flags, DIFFUSE_ENCODING_FLAG);
        ImGui::FlagCheckbox("Enable Self Learning", &renderer.params.flags, SELF_LEARNING_FLAG);
        ImGui::SliderFloat("Training Direction", &renderer.trainingDirection, 0.0f, 1.0f, "%.2f");
        ImGui::PlotLines("Loss", renderer.lossHistory.data(), renderer.lossHistory.size());
        if (ImGui::Button("Reset NRC")) {
            renderer.resetNRC();
            reset = true;
        }
    }

    if (ImGui::CollapsingHeader("Eye Training", ImGuiTreeNodeFlags_DefaultOpen)) {
        ImGui::FlagCheckbox("Russian Roulette##2", &renderer.params.flags, FORWARD_RR_FLAG);
    }

    if (ImGui::CollapsingHeader("Light Training", ImGuiTreeNodeFlags_DefaultOpen)) {
        ImGui::Combo("Training Mode", &renderer.backwardTrainer, {
            {TRAIN_LIGHT, "Light"},
            {TRAIN_LIGHT_NAIVE, "Light Naive"},
            {TRAIN_BIDIR, "Bidirectional"},
        });
        ImGui::FlagCheckbox("Russian Roulette##3", &renderer.params.flags, BACKWARD_RR_FLAG);
        float balancing = 100.0f - 100.0f / renderer.params.balanceWeight;
        if (ImGui::SliderFloat("Balancing Samples", &balancing, 0.0f, 100.0f, "%.0f%%")) {
            renderer.params.balanceWeight = 100.0f / (100.0f - balancing);
        }
        ImGui::Text("(Balancing Weight: %.2f)", renderer.params.balanceWeight);
        ImGui::SliderFloat("Photon Query Samples", &renderer.photonMappingAmount, 0.0f, 1.0f, "%.2f");
        ImGui::SliderFloat("Photon Query Replacement Factor", &renderer.photonQueryReplacement, 0.0f, 1.0f, "%.2f");
        ImGui::SliderInt("Photon Count", reinterpret_cast<int*>(&renderer.photonCount), 1 << 10, 1 << 20, "%d");
        ImGui::SliderFloat("Photon Radius", &renderer.sppmBVH.initialRadius, 0.01f, 1.0f, "%.2f");
        ImGui::SliderFloat("Alpha", &renderer.sppmBVH.alpha, 0.0f, 1.0f, "%.2f");
    }

    ImGui::End();

    if (reset) renderer.reset();
}

void MainApp::render() {
    // Map the buffer to CUDA
    vec4* image;
    size_t size;
    check(hipGraphicsMapResources(1, &cudaPboResource));
    check(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&image), &size, cudaPboResource));

    const float camDelta = 2.5f * delta;
    if (isKeyDown(Key::W)) camera.moveInEyeSpace(vec3(0.0f, 0.0f, -camDelta));
    if (isKeyDown(Key::S)) camera.moveInEyeSpace(vec3(0.0f, 0.0f, camDelta));
    if (isKeyDown(Key::A)) camera.moveInEyeSpace(vec3(-camDelta, 0.0f, 0.0f));
    if (isKeyDown(Key::D)) camera.moveInEyeSpace(vec3(camDelta, 0.0f, 0.0f));
    if (isKeyDown(Key::Q)) camera.moveInEyeSpace(vec3(0.0f, -camDelta, 0.0f));
    if (isKeyDown(Key::E)) camera.moveInEyeSpace(vec3(0.0f, camDelta, 0.0f));

    if(camera.updateIfChanged()) {
        renderer.setCamera(inverse(camera.projectionMatrix * camera.viewMatrix));
        renderer.reset();
    }

    renderer.render(image, bufferDim);

    // Unmap the buffer
    check(hipGraphicsUnmapResources(1, &cudaPboResource));

    // Map the buffer to a texture
    pbo.bind();
    glTextureSubImage2D(blitTexture.handle, 0, 0, 0, bufferDim.x, bufferDim.y, GL_RGBA, GL_FLOAT, nullptr);

    // Blit the texture using OpenGL
    fullscreenTriangle.draw();
}