#include "mainapp.cuh"

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <optix_types.h>

#include <framework/gl/buffer.hpp>
#include <framework/imguiutil.hpp>

#include <glm/glm.hpp>
#include <glm/gtc/type_ptr.hpp>
using namespace glm;

#include <imgui.h>
#include <misc/cpp/imgui_stdlib.h>

#include <iostream>
#include <format>
#include <filesystem>

#include "cudautil.hpp"

const std::string vs = R"(#version 460 core
layout(location = 0) in vec2 position;
out vec2 texCoord;
void main() {
    gl_Position = vec4(position, 0.0, 1.0);
    texCoord = position * 0.5 + 0.5;
})";

const std::string fs = R"(#version 460 core
in vec2 texCoord;
out vec4 fragColor;
layout(location = 0) uniform sampler2D tex;
layout(location = 1) uniform float exposure;
void main() {
    fragColor = texture(tex, texCoord) * exposure;
})";

std::vector<std::filesystem::path> scanFolder(const std::filesystem::path& folder) {
    std::vector<std::filesystem::path> files;
    try {
        for (auto& f : std::filesystem::directory_iterator(folder)) {
            if (f.path().extension() == ".glb"|| f.path().extension() == ".gltf") files.push_back(f.path());
        } 
        return files;
    } catch (const std::filesystem::filesystem_error& e) {
        std::cerr << e.what() << std::endl;
        return files;
    }
}

MainApp::MainApp() : App(800, 600) {
    printCudaDevices();

    fullscreenTriangle.load(Mesh::FULLSCREEN_VERTICES, Mesh::FULLSCREEN_INDICES);

    blitProgram.loadSource(vs, fs);
    blitProgram.use();
    blitProgram.set(1, exposure);

    folder = std::filesystem::current_path().parent_path().string();
    scenes = scanFolder(folder);

    setVSync(false);
}

MainApp::~MainApp() {
    check(hipGraphicsUnregisterResource(cudaPboResource));
}

void MainApp::resize(const ivec2& res) {
    bufferDim = uvec2(res);
    if (cudaPboResource) check(hipGraphicsUnregisterResource(cudaPboResource)); // Unregister the old resource to prevent memory leak
    pbo.allocate(res.x * res.y * sizeof(vec4), GL_STREAM_DRAW);
    check(hipGraphicsGLRegisterBuffer(&cudaPboResource, pbo.handle, cudaGraphicsMapFlagsWriteDiscard));
    blitTexture = Texture<GL_TEXTURE_2D>();
    blitTexture.allocate2D(GL_RGBA32F, res.x, res.y);
    blitTexture.bindTextureUnit(0);
    camera.resize(float(res.x) / float(res.y));
    renderer.reset();
    renderer.resize(uvec2(res));
}

void MainApp::resizeCallback(const vec2& res) { 
    resize(res); // TODO: Fix scaling issues
}

void MainApp::keyCallback(Key key, Action action, Modifier modifier) {
    if (action == Action::PRESS && key == Key::ESC) close();
}

void MainApp::scrollCallback(float xamount, float yamount) {
    camera.zoom(yamount);
}

void MainApp::moveCallback(const vec2& movement, bool leftButton, bool rightButton, bool middleButton) {
    if (leftButton) camera.orbit(movement * 0.01f);
}

void MainApp::buildImGui() {
    ImGui::StatisticsWindow(delta, resolution);

    ImGui::Begin("Settings", nullptr, ImGuiWindowFlags_AlwaysAutoResize);

    std::string folder_str = folder.string();
    if (ImGui::InputText("Folder", &folder_str)) {
        folder = folder_str;
        scenes = scanFolder(folder);
        sceneID = 0;
    }
    if (ImGui::FileCombo("Scene", &sceneID, scenes)) {
        renderer.loadGLTF(scenes.at(sceneID));
    }
    ImGui::Text("Sample: %d", renderer.params.sample);
    if (ImGui::SliderFloat("Exposure", &exposure, 0.1f, 10.0f, "%.1f", ImGuiSliderFlags_Logarithmic)) blitProgram.set(1, exposure);
    ImGui::SliderFloat("Russian Roulette", &renderer.params.russianRouletteWeight, 1.0f, 10.0f, "%.1f");
    ImGui::SliderFloat("Scene Epsilon", &renderer.params.sceneEpsilon, 1e-6f, 1e-1f, "%f", ImGuiSliderFlags_Logarithmic);
    bool reset = ImGui::FlagCheckbox("Enable NEE", &renderer.params.flags, NEE_FLAG);
    reset |= ImGui::FlagCheckbox("Enable Transmission", &renderer.params.flags, TRANSMISSION_FLAG);
    for (size_t i = 0; i < renderer.scene.cameras.size(); i++) {
        if (ImGui::Button(renderer.scene.cameras[i].first.c_str())) {
            auto scale = mat4(1.0f);
            scale[0][0] = camera.aspectRatio;
            const auto clipToWorld = renderer.scene.cameras[i].second * scale;
            renderer.setCamera(clipToWorld);
            reset = true;
        }
        if (i < renderer.scene.cameras.size() - 1) ImGui::SameLine();
    }

    ImGui::SeparatorText("NRC");
    ImGui::Checkbox("Enable Training", &renderer.enableTraining);
    ImGui::FlagCheckbox("Enable Backward RR", &renderer.params.flags, BACKWARD_RR_FLAG);
    ImGui::FlagCheckbox("Enable Forward RR", &renderer.params.flags, FORWARD_RR_FLAG);
    ImGui::SliderFloat("Training Direction", &renderer.trainingDirection, 0.0f, 1.0f, "%.2f");
    reset |= ImGui::EnumCombo("Inference Mode", &renderer.params.inferenceMode, INFERENCE_MODES);
    ImGui::PlotLines("Loss", renderer.lossHistory.data(), renderer.lossHistory.size());
    if (ImGui::Button("Reset NRC")) {
        renderer.resetNRC();
        reset = true;
    }

    ImGui::End();

    if (reset) renderer.reset();
}

void MainApp::render() {
    // Map the buffer to CUDA
    vec4* image;
    size_t size;
    check(hipGraphicsMapResources(1, &cudaPboResource));
    check(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&image), &size, cudaPboResource));

    if(camera.updateIfChanged()) {
        renderer.setCamera(inverse(camera.projectionMatrix * camera.viewMatrix));
        renderer.reset();
    }

    renderer.render(image, bufferDim);

    // Unmap the buffer
    check(hipGraphicsUnmapResources(1, &cudaPboResource));

    // Map the buffer to a texture
    pbo.bind();
    glTextureSubImage2D(blitTexture.handle, 0, 0, 0, bufferDim.x, bufferDim.y, GL_RGBA, GL_FLOAT, nullptr);

    // Blit the texture using OpenGL
    fullscreenTriangle.draw();
}