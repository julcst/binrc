#include "mainapp.cuh"

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <optix_types.h>

#include <framework/gl/buffer.hpp>
#include <framework/imguiutil.hpp>

#include <glm/glm.hpp>
#include <glm/gtc/type_ptr.hpp>
using namespace glm;

#include <imgui.h>
#include <misc/cpp/imgui_stdlib.h>

#include <iostream>
#include <format>
#include <filesystem>

#include "cudautil.hpp"

const std::string vs = R"(#version 460 core
layout(location = 0) in vec2 position;
out vec2 texCoord;
void main() {
    gl_Position = vec4(position, 0.0, 1.0);
    texCoord = position * 0.5 + 0.5;
})";

const std::string fs = R"(#version 460 core
in vec2 texCoord;
out vec4 fragColor;
layout(location = 0) uniform sampler2D tex;
layout(location = 1) uniform float exposure;
void main() {
    fragColor = texture(tex, texCoord) * exposure;
})";

std::vector<std::filesystem::path> scanFolder(const std::filesystem::path& folder) {
    std::vector<std::filesystem::path> files;
    try {
        for (auto& f : std::filesystem::directory_iterator(folder)) {
            if (f.path().extension() == ".glb"|| f.path().extension() == ".gltf") files.push_back(f.path());
        } 
        return files;
    } catch (const std::filesystem::filesystem_error& e) {
        std::cerr << e.what() << std::endl;
        return files;
    }
}

bool FileCombo(const char* label, size_t* curr, const std::vector<std::filesystem::path>& items) {
    return ImGui::Combo(
        label, reinterpret_cast<int*>(curr),
        [](void* data, int idx, const char** out_text) {
            auto items = reinterpret_cast<const std::vector<std::filesystem::path>*>(data);
            *out_text = items->at(idx).c_str();
            return true;
        },
        const_cast<void*>(reinterpret_cast<const void*>(&items)), items.size());
}

bool FlagCheckbox(const char* label, unsigned int* flags, unsigned int flag) {
    bool v = *flags & flag;
    bool changed = ImGui::Checkbox(label, &v);
    if (changed) {
        if (v) *flags |= flag;
        else *flags &= ~flag;
    }
    return changed;
}

MainApp::MainApp() : App(800, 600) {
    printCudaDevices();

    fullscreenTriangle.load(Mesh::FULLSCREEN_VERTICES, Mesh::FULLSCREEN_INDICES);

    blitProgram.loadSource(vs, fs);
    blitProgram.use();
    blitProgram.set(1, exposure);

    folder = std::filesystem::current_path().parent_path().string();
    scenes = scanFolder(folder);

    setVSync(false);
}

MainApp::~MainApp() {
    check(hipGraphicsUnregisterResource(cudaPboResource));
}

void MainApp::resizeCallback(const vec2& res) { 
    if (cudaPboResource) check(hipGraphicsUnregisterResource(cudaPboResource)); // Unregister the old resource to prevent memory leak
    pbo.allocate(res.x * res.y * sizeof(vec4), GL_STREAM_DRAW);
    check(hipGraphicsGLRegisterBuffer(&cudaPboResource, pbo.handle, cudaGraphicsMapFlagsWriteDiscard));
    blitTexture = Texture<GL_TEXTURE_2D>();
    blitTexture.allocate2D(GL_RGBA32F, res.x, res.y);
    blitTexture.bindTextureUnit(0);
    camera.resize(res.x / res.y);
    renderer.reset();
    renderer.resize(uvec2(res));
}

void MainApp::keyCallback(Key key, Action action, Modifier modifier) {
    if (action == Action::PRESS && key == Key::ESC) close();
}

void MainApp::scrollCallback(float amount) {
    camera.zoom(amount);
}

void MainApp::moveCallback(const vec2& movement, bool leftButton, bool rightButton, bool middleButton) {
    if (leftButton) camera.orbit(movement * 0.01f);
}

void MainApp::buildImGui() {
    ImGui::StatisticsWindow(delta, resolution);

    ImGui::Begin("Settings", nullptr, ImGuiWindowFlags_AlwaysAutoResize);
    std::string folder_str = folder.string();
    if (ImGui::InputText("Folder", &folder_str)) {
        folder = folder_str;
        scenes = scanFolder(folder);
        sceneID = 0;
    }
    if (FileCombo("Scene", &sceneID, scenes)) {
        renderer.loadGLTF(scenes.at(sceneID));
    }
    ImGui::Text("Sample: %d", renderer.params->sample);
    if (ImGui::SliderFloat("Exposure", &exposure, 0.1f, 10.0f, "%.1f", ImGuiSliderFlags_Logarithmic)) blitProgram.set(1, exposure);
    ImGui::SliderFloat("Russian Roulette", &renderer.params->russianRouletteWeight, 1.0f, 10.0f, "%.1f");
    ImGui::SliderFloat("Scene Epsilon", &renderer.params->sceneEpsilon, 1e-6f, 1e-1f, "%f", ImGuiSliderFlags_Logarithmic);
    bool reset = FlagCheckbox("Enable NEE", &renderer.params->flags, NEE_FLAG);
    reset |= FlagCheckbox("Enable Transmission", &renderer.params->flags, TRANSMISSION_FLAG);
    for (size_t i = 0; i < renderer.scene.cameras.size(); i++) {
        if (ImGui::Button(renderer.scene.cameras[i].first.c_str())) {
            auto scale = mat4(1.0f);
            scale[0][0] = camera.aspectRatio;
            const auto clipToWorld = renderer.scene.cameras[i].second * scale;
            renderer.setCamera(clipToWorld);
            reset = true;
        }
        ImGui::SameLine();
    }
    ImGui::SeparatorText("NRC");
    reset |= FlagCheckbox("Enable NRC Inference", &renderer.params->flags, NRC_INFERENCE_FLAG);
    ImGui::PlotLines("Loss", renderer.lossHistory.data(), renderer.lossHistory.size());
    if (ImGui::Button("Reset NRC")) renderer.resetNRC();
    ImGui::End();

    if (reset) renderer.reset();
}

void MainApp::render() {
    // Map the buffer to CUDA
    vec4* image;
    size_t size;
    check(hipGraphicsMapResources(1, &cudaPboResource));
    check(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&image), &size, cudaPboResource));
    const auto dim = uvec2(resolution);

    if(camera.updateIfChanged()) {
        renderer.setCamera(inverse(camera.projectionMatrix * camera.viewMatrix));
        renderer.reset();
    }

    renderer.render(image, dim);

    // Unmap the buffer
    check(hipGraphicsUnmapResources(1, &cudaPboResource));

    // Map the buffer to a texture
    pbo.bind();
    glTextureSubImage2D(blitTexture.handle, 0, 0, 0, dim.x, dim.y, GL_RGBA, GL_FLOAT, nullptr);

    // Blit the texture using OpenGL
    fullscreenTriangle.draw();
}