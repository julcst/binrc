#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <array>

#include "params.cuh"
#include "payload.cuh"
#include "common.cuh"
#include "nrc.cuh"
#include "cudamath.cuh"
#include "sampling.cuh"
#include "principled_brdf.cuh"

constexpr uint N_RANDS = 2 + (TRAIN_DEPTH) * 9;

extern "C" __global__ void __raygen__() {
    const auto idx = optixGetLaunchIndex();
    const auto dim = optixGetLaunchDimensions();
    const auto i = idx.y * params.dim.x + idx.x;
    
    hiprandStatePhilox4_32_10_t state;
    hiprand_init(0, i, params.trainingRound * N_RANDS, &state);

    auto ray = makeCameraRay({hiprand_uniform(&state), hiprand_uniform(&state)});

    const auto nee = params.lightTable && (params.flags & NEE_FLAG);

    Payload payload;
    auto throughput = make_float3(1.0f);
    auto lightPdfIsZero = true;
    auto trainBounceIdx = 0;
    SampleResult sample {ray.direction, make_float3(1.0f), 1.0f, true, true};

    std::array<TrainBounce, TRAIN_DEPTH> trainBounces;
    for (uint i = 0; i < TRAIN_DEPTH; i++) {
        trainBounces[i].isValid = false;
    }
    
    for (uint depth = 1; depth <= TRAIN_DEPTH; depth++) {
        trainBounceIdx = depth - 1;

        // Russian roulette
        if (params.flags & FORWARD_RR_FLAG) {
            const float pContinue = min(luminance(throughput) * params.russianRouletteWeight, 1.0f);
            if (hiprand_uniform(&state) >= pContinue) break;
            for (uint i = 0; i < trainBounceIdx; i++) {
                trainBounces[i].throughput /= pContinue;
            }
            throughput /= pContinue;
        }

        payload = trace(ray);

        if (isinf(payload.t)) {
            for (uint i = 0; i < trainBounceIdx; i++) {
                trainBounces[i].radiance += trainBounces[i].throughput * sample.throughput * payload.emission;
            }
            break; // Skybox
        }

        for (uint i = 0; i <= trainBounceIdx; i++) {
            trainBounces[i].throughput *= sample.throughput;
        }

        auto n = payload.normal;
        const auto wo = -ray.direction;
        const auto inside = dot(n, wo) < 0.0f;
        n = inside ? -n : n;
        const auto hitPoint = ray.origin + payload.t * ray.direction;
        const auto alpha = payload.roughness * payload.roughness;
        const auto metallic = payload.metallic;
        const auto baseColor = payload.baseColor; // baseColor

        if (luminance(payload.emission) > 0.0f) {
            auto weight = 1.0f;
            if (nee && !lightPdfIsZero) {
                // NOTE: Maybe calculating the prevBrdfPdf here only when necessary is faster
                const auto lightPdf = lightPdfUniform(wo, payload.t, n, payload.area);
                weight = balanceHeuristic(sample.pdf, lightPdf);
            }
            for (uint i = 0; i < trainBounceIdx; i++) {
                trainBounces[i].radiance += trainBounces[i].throughput * payload.emission * weight;
            }
        }

        // Next event estimation
        if (nee) {
            const auto sample = sampleLight(hiprand_uniform(&state), {hiprand_uniform(&state), hiprand_uniform(&state)}, hitPoint);
            const auto cosThetaS = dot(sample.wi, n);
            //if (abs(cosThetaS) > 0.0f && abs(sample.cosThetaL) > 0.0f) {
                const auto brdf = evalDisney(wo, sample.wi, n, baseColor, metallic, alpha, payload.transmission, inside);
                const auto surfacePoint = hitPoint + n * copysignf(params.sceneEpsilon, cosThetaS);
                const auto lightPoint = sample.position - sample.n * copysignf(params.sceneEpsilon, dot(sample.wi, sample.n));
                if (!brdf.isDirac && brdf.pdf > 0.0f && !traceOcclusion(surfacePoint, lightPoint)) {
                    const auto weight = balanceHeuristic(sample.pdf, brdf.pdf);
                    const auto weightedEmission = brdf.throughput * sample.emission * weight / sample.pdf;
                    for (uint i = 0; i <= trainBounceIdx; i++) {
                        trainBounces[i].radiance += trainBounces[i].throughput * weightedEmission;
                    }
                }
            //}
        }

        const auto r = hiprand_uniform4(&state);
        sample = sampleDisney(hiprand_uniform(&state), {r.x, r.y}, {r.z, r.w}, wo, n, inside, payload.baseColor, payload.metallic, alpha, payload.transmission);
        
        ray = Ray{hitPoint + n * copysignf(params.sceneEpsilon, dot(sample.direction, n)), sample.direction};
        lightPdfIsZero = sample.isDirac;
        for (uint i = 0; i <= trainBounceIdx; i++) {
            trainBounces[i].throughput *= sample.throughput;
        }
        throughput *= sample.throughput;

        const auto trainInput = encodeInput(hitPoint, !sample.isSpecular && (params.flags & DIFFUSE_ENCODING_FLAG) ? make_float3(NAN) : wo, n, payload);
        const auto trainIdx = pushNRCTrainInput(trainInput);
        const auto reflectanceFactorizationTerm = 1.0f / max(trainInput.diffuse + trainInput.specular, 1e-3f);
        trainBounces[trainBounceIdx].index = trainIdx;
        trainBounces[trainBounceIdx].reflectanceFactorizationTerm = reflectanceFactorizationTerm;
        trainBounces[trainBounceIdx].isValid = true;
    }

    // TODO: Keep 1/16 of learning paths unbiased from self-learning
    if (params.flags & SELF_LEARNING_FLAG) {
        params.selfLearningBounces[i] = trainBounces;
        for (uint j = 0; j < NRC_INPUT_SIZE; j++) {
            params.selfLearningQueries[i * NRC_INPUT_SIZE + j] = params.trainingInput[trainBounces[trainBounceIdx].index * NRC_INPUT_SIZE + j];
        }
    } else {
        for (uint i = 0; i < trainBounceIdx; i++) {
            writeNRCOutput(params.trainingTarget + trainBounces[i].index * NRC_OUTPUT_SIZE, trainBounces[i].radiance * trainBounces[i].reflectanceFactorizationTerm);
        }
    }
}