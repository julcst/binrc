#include "hip/hip_runtime.h"
#include <optix.h>
#include <hiprand/hiprand_kernel.h>

#include "params.cuh"
#include "sampling.cuh"
#include "common.cuh"
#include "principled_brdf.cuh"

extern "C" __global__ void __raygen__() {
    if (!params.lightTable) return; // Cannot sample light without lights
    
    const auto idx = optixGetLaunchIndex();
    const auto dim = optixGetLaunchDimensions();
    const auto i = idx.y * params.dim.x + idx.x;

    hiprandStatePhilox4_32_10_t state;
    constexpr uint32_t N_RANDS = 4 * MAX_BOUNCES + 4; // Number of random numbers to generate at once
    hiprand_init(0, i, params.trainingRound * N_RANDS, &state);

    const auto r = hiprand_uniform4(&state); // Note hiprand generates in (0, 1] not [0, 1)
    const auto lightSample = samplePhoton(hiprand_uniform(&state), make_float2(r.x, r.y), make_float2(r.z, r.w));
    auto ray = Ray{lightSample.position + lightSample.n * copysignf(params.sceneEpsilon, dot(lightSample.wo, lightSample.n)), lightSample.wo};
    auto flux = lightSample.emission;

    Payload payload;

    bool isCaustic = false;

    for (uint depth = 0; depth < 32; depth++) {
        const auto r = hiprand_uniform4(&state);

        // Russian roulette
        const float pContinue = min(luminance(flux) * params.russianRouletteWeight, 1.0f);
        if (r.x >= pContinue) break;
        flux /= pContinue;

        payload = trace(ray);

        if (isinf(payload.t)) break; // Skybox

        const auto wi = -ray.direction;
        const auto hitPoint = ray.origin + payload.t * ray.direction;
        const auto alpha = payload.roughness * payload.roughness;
        const auto mat = calcMaterialProperties(payload.baseColor, payload.metallic, alpha, payload.transmission);

        if (mat.isDiffuse()) {
            if (isCaustic) { // Only store caustic photons or with some probability
                // Store photon
                params.photonMap.recordPhoton({
                    .pos = hitPoint,
                    .wi = wi,
                    .flux = flux,
                });
            } else if (hiprand_uniform(&state) < params.photonMap.photonRecordingProbability) {
                // Store photon only with some probability
                params.photonMap.recordPhoton({
                    .pos = hitPoint,
                    .wi = wi,
                    .flux = flux / params.photonMap.photonRecordingProbability,
                });
            }
        }

        const auto sample = sampleDisney(r.y, {r.z, r.w}, {r.z, r.w}, wi, payload.normal, payload.baseColor, payload.metallic, alpha, payload.transmission);
        flux *= sample.throughput; // FIXME, this is BRDF * cosThetaO / pdf, not BRDF * cosThetaI / pdf
        //flux += payload.emission; // TODO: Is this correct?
        isCaustic = !mat.isDiffuse();
        //isCaustic = balanceHeuristic(sample.pdf, abs(dot(payload.normal, sample.direction)) * INV_PI) > 0.7f;

        if (!isfinite(sample.direction)) {
            printf("Warning: NaN sample direction in light pass: depth=%d alpha=%f transmission=%f metallic=%f\n", depth, alpha, payload.transmission, payload.metallic);
            break;
        }

        ray = Ray{hitPoint + payload.normal * copysignf(params.sceneEpsilon, dot(sample.direction, payload.normal)), sample.direction};
    }
}