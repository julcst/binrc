#include "hip/hip_runtime.h"
#include <optix.h>
#include <hiprand/hiprand_kernel.h>

#include "params.cuh"
#include "sampling.cuh"
#include "common.cuh"
#include "principled_brdf.cuh"

extern "C" __global__ void __raygen__() {
    if (!params.lightTable) return; // Cannot sample light without lights
    
    const auto idx = optixGetLaunchIndex();
    const auto dim = optixGetLaunchDimensions();
    const auto i = idx.y * params.dim.x + idx.x;

    hiprandStatePhilox4_32_10_t state;
    constexpr uint32_t N_RANDS = 64; // Number of random numbers to generate at once
    hiprand_init(0, i, params.trainingRound * N_RANDS, &state);

    const auto r = hiprand_uniform4(&state); // Note hiprand generates in (0, 1] not [0, 1)
    const auto lightSample = sampleLight(hiprand_uniform(&state), make_float2(r.x, r.y), make_float2(r.z, r.w));
    auto ray = Ray{lightSample.position + lightSample.n * copysignf(params.sceneEpsilon, dot(lightSample.wo, lightSample.n)), lightSample.wo};
    auto radiance = lightSample.emission * INV_PI;

    Payload payload;

    for (uint depth = 1; depth <= TRAIN_DEPTH; depth++) {
        const auto r = hiprand_uniform4(&state);

        // Russian roulette
        if (params.flags & BACKWARD_RR_FLAG) {
            const float pContinue = min(luminance(radiance) * params.russianRouletteWeight, 1.0f);
            if (r.x >= pContinue) break; // FIXME: use random numbers independent from sampling
            radiance /= pContinue;
        }

        payload = trace(ray);

        if (isinf(payload.t)) break; // Skybox

        const auto wi = -ray.direction;
        const auto hitPoint = ray.origin + payload.t * ray.direction;
        const auto alpha = payload.roughness * payload.roughness;

        const auto sample = sampleDisney(r.y, {r.z, r.w}, {r.z, r.w}, wi, payload.normal, payload.baseColor, payload.metallic, alpha, payload.transmission);

        params.photonMap.recordPhoton({
            .pos = hitPoint,
            .wi = wi,
            .flux = radiance,
        });

        radiance *= sample.throughput;

        ray = Ray{hitPoint + payload.normal * copysignf(params.sceneEpsilon, dot(sample.direction, payload.normal)), sample.direction};

        radiance += payload.emission;
    }
}