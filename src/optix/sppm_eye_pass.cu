#include "hip/hip_runtime.h"
#include <optix.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "params.cuh"
#include "payload.cuh"
#include "principled_brdf.cuh"
#include "common.cuh"

extern "C" __global__ void __raygen__() {
    const auto idx = optixGetLaunchIndex();
    const auto dim = optixGetLaunchDimensions();
    const auto i = idx.y * params.dim.x + idx.x;
    
    hiprandStatePhilox4_32_10_t state;
    constexpr uint32_t N_RANDS = 64;
    hiprand_init(0, i, params.trainingRound * N_RANDS, &state);

    auto ray = makeCameraRay({hiprand_uniform(&state), hiprand_uniform(&state)});

    float3 throughput = make_float3(1.0f);

    for (uint depth = 0; depth <= 6; depth++) {

        const auto r = hiprand_uniform4(&state);

        // Russian roulette
        const float pContinue = min(luminance(throughput) * params.russianRouletteWeight, 1.0f);
        if (r.x >= pContinue) break;
        throughput /= pContinue;

        const auto payload = trace(ray);

        if (isinf(payload.t)) break; // Skybox

        const auto wo = -ray.direction;
        const auto hitPoint = ray.origin + payload.t * ray.direction;
        const auto alpha = payload.roughness * payload.roughness;
        const auto mat = calcMaterialProperties(payload.baseColor, payload.metallic, alpha, payload.transmission);
        const auto sample = sampleDisney(r.y, {r.z, r.w}, {r.z, r.w}, wo, payload.normal, payload.baseColor, payload.metallic, alpha, payload.transmission);

        // TODO: Check pdf <= 1 / PI
        if (luminance(mat.albedo * (1 - mat.transmission)) > 1e-6f) {
            params.photonMap.store({
                .pos = hitPoint,
                .wo = wo,
                .n = payload.normal,
                .mat = mat,
                .radius = params.photonMap.initialRadius, // TODO: Radius reduction
                .totalPhotonCountAtBirth = params.photonMap.totalPhotonCount,
            });
        }
        
        ray = Ray{hitPoint + payload.normal * copysignf(params.sceneEpsilon, dot(sample.direction, payload.normal)), sample.direction};
        throughput *= sample.throughput;
    }
}

extern "C" __global__ void __raygen__full() {
    const auto idx = optixGetLaunchIndex();
    const auto dim = optixGetLaunchDimensions();
    const auto i = idx.y * params.dim.x + idx.x;
    if (i > params.photonMap.queryCount) return; // Safety check

    hiprandStatePhilox4_32_10_t state;
    constexpr uint32_t N_RANDS = 64;
    hiprand_init(0, i, params.trainingRound * N_RANDS, &state);

    const auto uv = (make_float2(idx.x, idx.y) + make_float2(hiprand_uniform(&state), hiprand_uniform(&state))) / make_float2(dim.x, dim.y);
    auto ray = makeCameraRay(uv);

    float3 throughput = make_float3(1.0f);

    // TODO: Handle too long paths
    for (uint depth = 0; depth < 6; depth++) {
        const auto r = hiprand_uniform4(&state);

        const auto payload = trace(ray);

        if (isinf(payload.t)) {// Skybox => No photon query
            params.photonMap.store(i, {
                .radius = 0.0f, // Invalid query no radius
            });
            params.image[i] = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
            break;
        }

        const auto wo = -ray.direction;
        const auto hitPoint = ray.origin + payload.t * ray.direction;
        const auto alpha = payload.roughness * payload.roughness;
        const auto mat = calcMaterialProperties(payload.baseColor, payload.metallic, alpha, payload.transmission);
        const auto sample = sampleDisney(r.y, {r.z, r.w}, {r.z, r.w}, wo, payload.normal, payload.baseColor, payload.metallic, alpha, payload.transmission);

        // if (luminance(mat.albedo * (1 - mat.transmission)) > 1e-6f) {
        if (sample.pdf <= INV_PI + 1e-2f) {
            params.photonMap.store(i, {
                .pos = hitPoint,
                .wo = wo,
                .n = payload.normal,
                .mat = mat,
                .radius = params.photonMap.initialRadius, // TODO: Radius reduction
                .totalPhotonCountAtBirth = params.photonMap.totalPhotonCount,
            });
            params.image[i] = make_float4(throughput, 1.0f);
            break;
        }
        
        ray = Ray{hitPoint + payload.normal * copysignf(params.sceneEpsilon, dot(sample.direction, payload.normal)), sample.direction};
        throughput *= sample.throughput;
    }
}