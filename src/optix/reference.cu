#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <hip/hip_runtime.h>

#include "params.cuh"
#include "payload.cuh"
#include "common.cuh"
#include "cudamath.cuh"
#include "sampling.cuh"

extern "C" __global__ void __raygen__reference() {
    const auto idx = optixGetLaunchIndex();
    const auto dim = optixGetLaunchDimensions();
    const auto i = idx.y * params.dim.x + idx.x;
    const auto rotation = params.rotationTable[i];

    const auto uv = (make_float2(idx.x, idx.y) + RND_JITTER) / make_float2(dim.x, dim.y);
    auto ray = makeCameraRay(uv);

    const auto nee = params.lightTable && (params.flags & NEE_FLAG);

    Payload payload;
    auto color = make_float3(0.0f);
    auto throughput = make_float3(1.0f);
    auto prevBrdfPdf = 1.0f;
    auto lightPdfIsZero = true;
    
    for (uint depth = 1; depth < MAX_BOUNCES; depth++) {
        // Russian roulette
        const float pContinue = min(luminance(throughput) * params.russianRouletteWeight, 1.0f);
        if (RND_ROULETTE >= pContinue) break;
        throughput /= pContinue;

        payload = trace(ray);

        if (isinf(payload.t)) {
            color += throughput * payload.emission;
            break; // Skybox
        }

        auto n = payload.normal;
        const auto wo = -ray.direction;
        const auto inside = dot(n, wo) < 0.0f;
        n = inside ? -n : n;

        if (luminance(payload.emission) > 0.0f) {
            auto weight = 1.0f;
            if (nee && !lightPdfIsZero) {
                // NOTE: Maybe calculating the prevBrdfPdf here only when necessary is faster
                const auto lightPdf = lightPdfUniform(wo, payload.t, n, payload.area);
                weight = balanceHeuristic(prevBrdfPdf, lightPdf);
            }
            color += throughput * payload.emission * weight;
        }

        const auto hitPoint = ray.origin + payload.t * ray.direction;
        const auto alpha = payload.roughness * payload.roughness;
        const auto metallic = payload.metallic;
        const auto baseColor = payload.baseColor; // baseColor

        // Next event estimation
        if (nee) {
            const auto sample = sampleLight(RND_LSRC, RND_LSAMP, hitPoint);
            const auto cosThetaS = dot(sample.wi, n);
            //if (abs(cosThetaS) > 0.0f && abs(sample.cosThetaL) > 0.0f) {
                const auto brdf = evalDisney(wo, sample.wi, n, baseColor, metallic, alpha, payload.transmission, inside);
                const auto surfacePoint = hitPoint + n * copysignf(params.sceneEpsilon, cosThetaS);
                const auto lightPoint = sample.position - sample.n * copysignf(params.sceneEpsilon, dot(sample.wi, sample.n));
                if (!brdf.isDirac && brdf.pdf > 0.0f && !traceOcclusion(surfacePoint, lightPoint)) {
                    const auto weight = balanceHeuristic(sample.pdf, brdf.pdf);
                    color += throughput * brdf.throughput * sample.emission * weight / sample.pdf;
                }
            //}
        }

        const auto sample = sampleDisney(RND_BSDF, RND_MICROFACET, RND_DIFFUSE, wo, n, inside, payload.baseColor, payload.metallic, alpha, payload.transmission);
        
        ray = Ray{hitPoint + n * copysignf(params.sceneEpsilon, dot(sample.direction, n)), sample.direction};
        throughput *= sample.throughput;
        prevBrdfPdf = sample.pdf;
        lightPdfIsZero = sample.isDirac || payload.transmission > 0.0f;
    }

    params.image[i] = mix(params.image[i], make_float4(max(color, 0.0f), 1.0f), params.weight);
}